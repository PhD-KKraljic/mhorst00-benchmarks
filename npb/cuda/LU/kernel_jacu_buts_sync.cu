#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is an CUDA® C version of the NPB LU code. This CUDA® C  //
//  version is a part of SNU-NPB 2019 developed by the Center for Manycore //
//  Programming at Seoul National University and derived from the serial   //
//  Fortran versions in "NPB3.3.1-SER" developed by NAS.                   //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on original NPB 3.3.1, including the technical report, the //
//  original specifications, source code, results and information on how   //
//  to submit new results, is available at:                                //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Information on SNU-NPB 2019, including the conference paper and source //
//  code, is available at:                                                 //
//                                                                         //
//           http://aces.snu.ac.kr                                         //
//                                                                         //
//  Send comments or suggestions for this CUDA® C version to               //
//  snunpb@aces.snu.ac.kr                                                  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 08826, Korea                                             //
//                                                                         //
//          E-mail: snunpb@aces.snu.ac.kr                                  //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Youngdong Do, Hyung Mo Kim, Pyeongseok Oh, Daeyoung Park,      //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include <stdio.h>
#include <math.h>
#include "applu.incl"
#include "kernel_constants.h"

extern "C" {
#include "timers.h"
}



__global__
void k_jbu_datagen_sync(double *m_u, 
                        double *m_qs,
                        double *m_rho_i, 
                        int jst, int jend, 
                        int ist, int iend,
                        int temp_kst, 
                        int temp_kend, 
                        int work_num_item)
{
  int k = blockDim.z * blockIdx.z + threadIdx.z;
  int j = blockDim.y * blockIdx.y + threadIdx.y + jst;
  int i = blockDim.x * blockIdx.x + threadIdx.x + ist;

  if (k > work_num_item || j > jend || i > iend) return;

  k += temp_kst;

  double tmp;

  double (* u)[ISIZ2/2*2+1][ISIZ1/2*2+1][5]
    = (double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5])m_u;
  double (* qs)[ISIZ2/2*2+1][ISIZ1/2*2+1]
    = (double (*) [ISIZ2/2*2+1][ISIZ1/2*2+1])m_qs;
  double (* rho_i)[ISIZ2/2*2+1][ISIZ1/2*2+1]
    = (double (*) [ISIZ2/2*2+1][ISIZ1/2*2+1])m_rho_i;


  tmp = 1.0 / u[k][j][i][0];
  rho_i[k][j][i] = tmp;
  qs[k][j][i] = 0.50 * (  u[k][j][i][1] * u[k][j][i][1]
      + u[k][j][i][2] * u[k][j][i][2]
      + u[k][j][i][3] * u[k][j][i][3] ) * tmp;


}

__global__
void k_buts_BR_sync(double *m_v, 
                    double *m_rho_i,
                    double *m_u, 
                    double *m_qs,
                    double *m_au,
                    double *m_bu,
                    double *m_cu,
                    double *m_du,
                    int kst, int kend, 
                    int jst, int jend, 
                    int ist, int iend, 
                    int wg00_tail_k, 
                    int wg00_tail_j, 
                    int wg00_tail_i,
                    int wg00_block_k,
                    int wg00_block_j, 
                    int wg00_block_i, 
                    int num_block_k, 
                    int num_block_j, 
                    int num_block_i, 
                    int block_size,
                    int block_size_k)
{
  int wg_tail_k, wg_tail_j, wg_tail_i;
  int wg_id = blockIdx.x;
  int l_j, l_i;

  int k, j, i, m;
  int cur_diag_blocks;
  int iter, step, dummy = 0;  
  int id, diag, head, diag_tmp;

  double tmp, tmp1; 

  // pointer casting
  double (* v)[ISIZ2/2*2+1][ISIZ1/2*2+1][5]
    = (double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5])m_v;

  double (* au)[5];
  double (* bu)[5];
  double (* cu)[5];
  double (* du)[5];

  double tv[5];
  double tmat[5][5];
  double temp_v[5];

  wg_tail_k = wg00_tail_k;
  wg_tail_j = wg00_tail_j;
  wg_tail_i = wg00_tail_i;


  // get current work_groups_head
  while(1) {
    cur_diag_blocks = min(wg00_block_j+1, num_block_i - wg00_block_i);
    if (cur_diag_blocks > wg_id) {
      wg_tail_j += wg_id*block_size;
      wg_tail_i -= wg_id*block_size;
      break;
    }
    else {
      if (wg00_block_j < num_block_j) {
        wg00_block_j++;
        wg_tail_j -= block_size;

        if (wg00_block_j >= num_block_j) {
          wg00_block_j--;
          wg00_block_i++;
          wg_tail_j += block_size;
          wg_tail_i -= block_size;
        }
      }
      else {
        wg00_block_i++;
        wg_tail_i -= block_size;
      }
      wg_tail_k += block_size_k;
      wg_id -= cur_diag_blocks;

    }
  }
  wg_id = blockIdx.x;


  // remapping work item order - work item 0 calculates the last element 
  id = block_size*block_size-1 - threadIdx.x;


  if (id < block_size*(block_size + 1) / 2) {
    diag = floor( ( sqrt((float)(1+8*id)) - 1 ) / 2);
    head = diag*(diag + 1) / 2;
    l_j = diag - (id - head);
    l_i = id - head;
  }
  else {
    diag = 2*block_size-1 - floor((sqrt((float)(1+8*(block_size*block_size-1 - id)))-1)/2) - 1;

    diag_tmp = 2*block_size-2 - diag;
    head = block_size*block_size - 1 - diag_tmp*(diag_tmp + 1) / 2;

    l_i = (block_size - 1) - (head - id);
    l_j = (block_size - 1) - diag_tmp + head - id;
  }

  k = wg_tail_k + 2*block_size - 2 - (l_j + l_i);
  j = wg_tail_j - (block_size - l_j - 1);
  i = wg_tail_i - (block_size - l_i - 1);



  dummy = 0;
  iter = min(wg_tail_k - kst + 1, block_size_k);
  iter += min(wg_tail_j - jst + 1, block_size);
  iter += min(wg_tail_i - ist + 1, block_size);
  iter -= 2;

  int t_diag, t_id, tail;

  for (step = 0; step < iter; step++) {

    dummy = 0;

    // work item remapping to active cells
    if (step >= block_size_k) {
      // same as jacld_blts
      // get id of wg0's item
      t_diag = step - block_size_k + 1;

      if (t_diag <= block_size) {
        t_id = t_diag*(t_diag+1)/2; 
      }
      else {
        diag_tmp = 2*block_size-1 - t_diag;
        t_id = block_size*block_size - diag_tmp*(diag_tmp+1)/2;
      }

      //id = get_local_id(0) + t_id;
      id = threadIdx.x + t_id;

      if (id < block_size*(block_size+1)/2) {
        diag = floor((sqrt((float)(1+8*id))-1)/2);
        tail = diag*(diag + 1) / 2;
        l_j = diag - (id - tail);
        l_i = id - tail;
      }
      else {
        diag = 2*block_size-1 - floor((sqrt((float)(1+8*(block_size*block_size-1 - id)))-1)/2) - 1;

        diag_tmp = 2*block_size-2 - diag;
        tail = block_size*block_size - 1 - diag_tmp*(diag_tmp + 1) / 2;

        l_i = (block_size - 1) - (tail - id);
        l_j = (block_size - 1) - diag_tmp + tail - id;
      }

      // opposite with jacld_blts
      l_j = block_size-1 - l_j;
      l_i = block_size-1 - l_i;

      t_diag = 2*block_size - 2 - t_diag;

      id = block_size*block_size-1 - id;

      k = wg_tail_k - block_size_k+1 + (t_diag - (l_i+l_j));
      j = wg_tail_j - (block_size-1 - l_j);
      i = wg_tail_i - (block_size-1 - l_i);


    }        
    else {
      k = wg_tail_k + 2*block_size -2 - (l_j+l_i) - step;
    }


    if (k <= wg_tail_k-block_size_k || k < kst || k > wg_tail_k || k >= kend || j < jst || i < ist) dummy = 1;
    if (id < 0 || id >= block_size*block_size) dummy = 1;

    if (!dummy) {
      au = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_au)[k][j][i];
      bu = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_bu)[k][j][i];
      cu = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_cu)[k][j][i];
      du = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_du)[k][j][i];

      for (m = 0; m < 5; m++) temp_v[m] = v[k][j][i][m];
      //#####################################################
      //#####################################################
      //          PART 1 - 1
      //#####################################################
      //#####################################################

      for(m = 0; m < 5; m++){
        tv[m] = 
          omega * (  cu[0][m] * v[k+1][j][i][0]
              + cu[1][m] * v[k+1][j][i][1]
              + cu[2][m] * v[k+1][j][i][2]
              + cu[3][m] * v[k+1][j][i][3]
              + cu[4][m] * v[k+1][j][i][4] );
      }









      //#####################################################
      //#####################################################
      //          PART 1 - 2
      //#####################################################
      //#####################################################

      for (m = 0; m < 5; m++) {
        tv[m] = tv[m]
          + omega * ( au[0][m] * v[k][j][i+1][0]
              + au[1][m] * v[k][j][i+1][1]
              + au[2][m] * v[k][j][i+1][2]
              + au[3][m] * v[k][j][i+1][3]
              + au[4][m] * v[k][j][i+1][4] 
              );
      }









      //#####################################################
      //#####################################################
      //          PART 1 - 3
      //#####################################################
      //#####################################################


      for (m = 0; m < 5; m++) {
        tv[m] += omega * ( bu[0][m] * v[k][j+1][i][0]
            + bu[1][m] * v[k][j+1][i][1]
            + bu[2][m] * v[k][j+1][i][2]
            + bu[3][m] * v[k][j+1][i][3]
            + bu[4][m] * v[k][j+1][i][4]
            );
      }


      //#####################################################
      //#####################################################
      //          PART 1 - 4
      //#####################################################
      //#####################################################

      //---------------------------------------------------------------------
      // diagonal block inversion
      //---------------------------------------------------------------------
      for (m = 0; m < 5; m++) {
        tmat[m][0] = du[0][m];
        tmat[m][1] = du[1][m];
        tmat[m][2] = du[2][m];
        tmat[m][3] = du[3][m];
        tmat[m][4] = du[4][m];
      }

      tmp1 = 1.0 / tmat[0][0];
      tmp = tmp1 * tmat[1][0];
      tmat[1][1] =  tmat[1][1] - tmp * tmat[0][1];
      tmat[1][2] =  tmat[1][2] - tmp * tmat[0][2];
      tmat[1][3] =  tmat[1][3] - tmp * tmat[0][3];
      tmat[1][4] =  tmat[1][4] - tmp * tmat[0][4];
      tv[1] = tv[1] - tv[0] * tmp;

      tmp = tmp1 * tmat[2][0];
      tmat[2][1] =  tmat[2][1] - tmp * tmat[0][1];
      tmat[2][2] =  tmat[2][2] - tmp * tmat[0][2];
      tmat[2][3] =  tmat[2][3] - tmp * tmat[0][3];
      tmat[2][4] =  tmat[2][4] - tmp * tmat[0][4];
      tv[2] = tv[2] - tv[0] * tmp;

      tmp = tmp1 * tmat[3][0];
      tmat[3][1] =  tmat[3][1] - tmp * tmat[0][1];
      tmat[3][2] =  tmat[3][2] - tmp * tmat[0][2];
      tmat[3][3] =  tmat[3][3] - tmp * tmat[0][3];
      tmat[3][4] =  tmat[3][4] - tmp * tmat[0][4];
      tv[3] = tv[3] - tv[0] * tmp;

      tmp = tmp1 * tmat[4][0];
      tmat[4][1] =  tmat[4][1] - tmp * tmat[0][1];
      tmat[4][2] =  tmat[4][2] - tmp * tmat[0][2];
      tmat[4][3] =  tmat[4][3] - tmp * tmat[0][3];
      tmat[4][4] =  tmat[4][4] - tmp * tmat[0][4];
      tv[4] = tv[4] - tv[0] * tmp;

      tmp1 = 1.0 / tmat[1][1];
      tmp = tmp1 * tmat[2][1];
      tmat[2][2] =  tmat[2][2] - tmp * tmat[1][2];
      tmat[2][3] =  tmat[2][3] - tmp * tmat[1][3];
      tmat[2][4] =  tmat[2][4] - tmp * tmat[1][4];
      tv[2] = tv[2] - tv[1] * tmp;

      tmp = tmp1 * tmat[3][1];
      tmat[3][2] =  tmat[3][2] - tmp * tmat[1][2];
      tmat[3][3] =  tmat[3][3] - tmp * tmat[1][3];
      tmat[3][4] =  tmat[3][4] - tmp * tmat[1][4];
      tv[3] = tv[3] - tv[1] * tmp;

      tmp = tmp1 * tmat[4][1];
      tmat[4][2] =  tmat[4][2] - tmp * tmat[1][2];
      tmat[4][3] =  tmat[4][3] - tmp * tmat[1][3];
      tmat[4][4] =  tmat[4][4] - tmp * tmat[1][4];
      tv[4] = tv[4] - tv[1] * tmp;

      tmp1 = 1.0 / tmat[2][2];
      tmp = tmp1 * tmat[3][2];
      tmat[3][3] =  tmat[3][3] - tmp * tmat[2][3];
      tmat[3][4] =  tmat[3][4] - tmp * tmat[2][4];
      tv[3] = tv[3] - tv[2] * tmp;

      tmp = tmp1 * tmat[4][2];
      tmat[4][3] =  tmat[4][3] - tmp * tmat[2][3];
      tmat[4][4] =  tmat[4][4] - tmp * tmat[2][4];
      tv[4] = tv[4] - tv[2] * tmp;

      tmp1 = 1.0 / tmat[3][3];
      tmp = tmp1 * tmat[4][3];
      tmat[4][4] =  tmat[4][4] - tmp * tmat[3][4];
      tv[4] = tv[4] - tv[3] * tmp;

      //---------------------------------------------------------------------
      // back substitution
      //---------------------------------------------------------------------
      tv[4] = tv[4] / tmat[4][4];

      tv[3] = tv[3] - tmat[3][4] * tv[4];
      tv[3] = tv[3] / tmat[3][3];

      tv[2] = tv[2]
        - tmat[2][3] * tv[3]
        - tmat[2][4] * tv[4];
      tv[2] = tv[2] / tmat[2][2];

      tv[1] = tv[1]
        - tmat[1][2] * tv[2]
        - tmat[1][3] * tv[3]
        - tmat[1][4] * tv[4];
      tv[1] = tv[1] / tmat[1][1];

      tv[0] = tv[0]
        - tmat[0][1] * tv[1]
        - tmat[0][2] * tv[2]
        - tmat[0][3] * tv[3]
        - tmat[0][4] * tv[4];
      tv[0] = tv[0] / tmat[0][0];


      temp_v[0] = temp_v[0] - tv[0];
      temp_v[1] = temp_v[1] - tv[1];
      temp_v[2] = temp_v[2] - tv[2];
      temp_v[3] = temp_v[3] - tv[3];
      temp_v[4] = temp_v[4] - tv[4];


    }


    if (!dummy) {
      for (m = 0; m < 5; m++) {
        v[k][j][i][m] = temp_v[m];
      }

    }

    __syncthreads();

  }

}

__global__
void k_jacu_sync(double *m_rsd,
                 double *m_u,
                 double *m_qs,
                 double *m_rho_i,
                 double *m_au,
                 double *m_bu,
                 double *m_cu,
                 double *m_du,
                 int nz, int ny, int nx,
                 int jst, int jend, 
                 int ist, int iend, 
                 int temp_kst, int temp_kend)
{
  int k = blockDim.z * blockIdx.z + threadIdx.z + temp_kst;
  int j = blockDim.y * blockIdx.y + threadIdx.y + jst;
  int i = blockDim.x * blockIdx.x + threadIdx.x + ist;

  double r43, c1345, c34,
         tmp1, tmp2, tmp3;

  double (*u)[ISIZ2/2*2+1][ISIZ1/2*2+1][5];
  double (*qs)[ISIZ2/2*2+1][ISIZ1/2*2+1];
  double (*rho_i)[ISIZ2/2*2+1][ISIZ1/2*2+1];

  double (*au)[ISIZ1/2*2+1][5][5];
  double (*bu)[ISIZ1/2*2+1][5][5];
  double (*cu)[ISIZ1/2*2+1][5][5];
  double (*du)[ISIZ1/2*2+1][5][5];

  if (k >= temp_kend || j >= jend || i >= iend)
    return;

  u = (double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5])m_u;
  qs = (double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1])m_qs;
  rho_i = (double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1])m_rho_i;

  au = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_au)[k];
  bu = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_bu)[k];
  cu = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_cu)[k];
  du = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_du)[k];

  r43 = ( 4.0 / 3.0 );
  c1345 = C1 * C3 * C4 * C5;
  c34 = C3 * C4;

  //---------------------------------------------------------------------
  // form the block daigonal
  //---------------------------------------------------------------------
  tmp1 = rho_i[k][j][i];
  tmp2 = tmp1 * tmp1;
  tmp3 = tmp1 * tmp2;

  du[j][i][0][0] = 1.0 + dt * 2.0 * ( tx1 * dx1 + ty1 * dy1 + tz1 * dz1 );
  du[j][i][1][0] = 0.0;
  du[j][i][2][0] = 0.0;
  du[j][i][3][0] = 0.0;
  du[j][i][4][0] = 0.0;

  du[j][i][0][1] =  dt * 2.0
    * ( - tx1 * r43 - ty1 - tz1 )
    * ( c34 * tmp2 * u[k][j][i][1] );
  du[j][i][1][1] =  1.0
    + dt * 2.0 * c34 * tmp1 
    * (  tx1 * r43 + ty1 + tz1 )
    + dt * 2.0 * ( tx1 * dx2 + ty1 * dy2 + tz1 * dz2 );
  du[j][i][2][1] = 0.0;
  du[j][i][3][1] = 0.0;
  du[j][i][4][1] = 0.0;

  du[j][i][0][2] = dt * 2.0
    * ( - tx1 - ty1 * r43 - tz1 )
    * ( c34 * tmp2 * u[k][j][i][2] );
  du[j][i][1][2] = 0.0;
  du[j][i][2][2] = 1.0
    + dt * 2.0 * c34 * tmp1
    * (  tx1 + ty1 * r43 + tz1 )
    + dt * 2.0 * ( tx1 * dx3 + ty1 * dy3 + tz1 * dz3 );
  du[j][i][3][2] = 0.0;
  du[j][i][4][2] = 0.0;

  du[j][i][0][3] = dt * 2.0
    * ( - tx1 - ty1 - tz1 * r43 )
    * ( c34 * tmp2 * u[k][j][i][3] );
  du[j][i][1][3] = 0.0;
  du[j][i][2][3] = 0.0;
  du[j][i][3][3] = 1.0
    + dt * 2.0 * c34 * tmp1
    * (  tx1 + ty1 + tz1 * r43 )
    + dt * 2.0 * ( tx1 * dx4 + ty1 * dy4 + tz1 * dz4 );
  du[j][i][4][3] = 0.0;

  du[j][i][0][4] = -dt * 2.0
    * ( ( ( tx1 * ( r43*c34 - c1345 )
            + ty1 * ( c34 - c1345 )
            + tz1 * ( c34 - c1345 ) ) * ( u[k][j][i][1]*u[k][j][i][1] )
          + ( tx1 * ( c34 - c1345 )
            + ty1 * ( r43*c34 - c1345 )
            + tz1 * ( c34 - c1345 ) ) * ( u[k][j][i][2]*u[k][j][i][2] )
          + ( tx1 * ( c34 - c1345 )
            + ty1 * ( c34 - c1345 )
            + tz1 * ( r43*c34 - c1345 ) ) * (u[k][j][i][3]*u[k][j][i][3])
        ) * tmp3
        + ( tx1 + ty1 + tz1 ) * c1345 * tmp2 * u[k][j][i][4] );

  du[j][i][1][4] = dt * 2.0
    * ( tx1 * ( r43*c34 - c1345 )
        + ty1 * (     c34 - c1345 )
        + tz1 * (     c34 - c1345 ) ) * tmp2 * u[k][j][i][1];
  du[j][i][2][4] = dt * 2.0
    * ( tx1 * ( c34 - c1345 )
        + ty1 * ( r43*c34 -c1345 )
        + tz1 * ( c34 - c1345 ) ) * tmp2 * u[k][j][i][2];
  du[j][i][3][4] = dt * 2.0
    * ( tx1 * ( c34 - c1345 )
        + ty1 * ( c34 - c1345 )
        + tz1 * ( r43*c34 - c1345 ) ) * tmp2 * u[k][j][i][3];
  du[j][i][4][4] = 1.0
    + dt * 2.0 * ( tx1 + ty1 + tz1 ) * c1345 * tmp1
    + dt * 2.0 * ( tx1 * dx5 + ty1 * dy5 + tz1 * dz5 );

  //---------------------------------------------------------------------
  // form the first block sub-diagonal
  //---------------------------------------------------------------------
  tmp1 = rho_i[k][j][i+1];
  tmp2 = tmp1 * tmp1;
  tmp3 = tmp1 * tmp2;

  au[j][i][0][0] = - dt * tx1 * dx1;
  au[j][i][1][0] =   dt * tx2;
  au[j][i][2][0] =   0.0;
  au[j][i][3][0] =   0.0;
  au[j][i][4][0] =   0.0;

  au[j][i][0][1] =  dt * tx2
    * ( - ( u[k][j][i+1][1] * tmp1 ) * ( u[k][j][i+1][1] * tmp1 )
        + C2 * qs[k][j][i+1] * tmp1 )
    - dt * tx1 * ( - r43 * c34 * tmp2 * u[k][j][i+1][1] );
  au[j][i][1][1] =  dt * tx2
    * ( ( 2.0 - C2 ) * ( u[k][j][i+1][1] * tmp1 ) )
    - dt * tx1 * ( r43 * c34 * tmp1 )
    - dt * tx1 * dx2;
  au[j][i][2][1] =  dt * tx2
    * ( - C2 * ( u[k][j][i+1][2] * tmp1 ) );
  au[j][i][3][1] =  dt * tx2
    * ( - C2 * ( u[k][j][i+1][3] * tmp1 ) );
  au[j][i][4][1] =  dt * tx2 * C2 ;

  au[j][i][0][2] =  dt * tx2
    * ( - ( u[k][j][i+1][1] * u[k][j][i+1][2] ) * tmp2 )
    - dt * tx1 * ( - c34 * tmp2 * u[k][j][i+1][2] );
  au[j][i][1][2] =  dt * tx2 * ( u[k][j][i+1][2] * tmp1 );
  au[j][i][2][2] =  dt * tx2 * ( u[k][j][i+1][1] * tmp1 )
    - dt * tx1 * ( c34 * tmp1 )
    - dt * tx1 * dx3;
  au[j][i][3][2] = 0.0;
  au[j][i][4][2] = 0.0;

  au[j][i][0][3] = dt * tx2
    * ( - ( u[k][j][i+1][1]*u[k][j][i+1][3] ) * tmp2 )
    - dt * tx1 * ( - c34 * tmp2 * u[k][j][i+1][3] );
  au[j][i][1][3] = dt * tx2 * ( u[k][j][i+1][3] * tmp1 );
  au[j][i][2][3] = 0.0;
  au[j][i][3][3] = dt * tx2 * ( u[k][j][i+1][1] * tmp1 )
    - dt * tx1 * ( c34 * tmp1 )
    - dt * tx1 * dx4;
  au[j][i][4][3] = 0.0;

  au[j][i][0][4] = dt * tx2
    * ( ( C2 * 2.0 * qs[k][j][i+1]
          - C1 * u[k][j][i+1][4] )
        * ( u[k][j][i+1][1] * tmp2 ) )
    - dt * tx1
    * ( - ( r43*c34 - c1345 ) * tmp3 * ( u[k][j][i+1][1]*u[k][j][i+1][1] )
        - (     c34 - c1345 ) * tmp3 * ( u[k][j][i+1][2]*u[k][j][i+1][2] )
        - (     c34 - c1345 ) * tmp3 * ( u[k][j][i+1][3]*u[k][j][i+1][3] )
        - c1345 * tmp2 * u[k][j][i+1][4] );
  au[j][i][1][4] = dt * tx2
    * ( C1 * ( u[k][j][i+1][4] * tmp1 )
        - C2
        * ( u[k][j][i+1][1]*u[k][j][i+1][1] * tmp2
          + qs[k][j][i+1] * tmp1 ) )
    - dt * tx1
    * ( r43*c34 - c1345 ) * tmp2 * u[k][j][i+1][1];
  au[j][i][2][4] = dt * tx2
    * ( - C2 * ( u[k][j][i+1][2]*u[k][j][i+1][1] ) * tmp2 )
    - dt * tx1
    * (  c34 - c1345 ) * tmp2 * u[k][j][i+1][2];
  au[j][i][3][4] = dt * tx2
    * ( - C2 * ( u[k][j][i+1][3]*u[k][j][i+1][1] ) * tmp2 )
    - dt * tx1
    * (  c34 - c1345 ) * tmp2 * u[k][j][i+1][3];
  au[j][i][4][4] = dt * tx2
    * ( C1 * ( u[k][j][i+1][1] * tmp1 ) )
    - dt * tx1 * c1345 * tmp1
    - dt * tx1 * dx5;

  //---------------------------------------------------------------------
  // form the second block sub-diagonal
  //---------------------------------------------------------------------
  tmp1 = rho_i[k][j+1][i];
  tmp2 = tmp1 * tmp1;
  tmp3 = tmp1 * tmp2;

  bu[j][i][0][0] = - dt * ty1 * dy1;
  bu[j][i][1][0] =   0.0;
  bu[j][i][2][0] =  dt * ty2;
  bu[j][i][3][0] =   0.0;
  bu[j][i][4][0] =   0.0;

  bu[j][i][0][1] =  dt * ty2
    * ( - ( u[k][j+1][i][1]*u[k][j+1][i][2] ) * tmp2 )
    - dt * ty1 * ( - c34 * tmp2 * u[k][j+1][i][1] );
  bu[j][i][1][1] =  dt * ty2 * ( u[k][j+1][i][2] * tmp1 )
    - dt * ty1 * ( c34 * tmp1 )
    - dt * ty1 * dy2;
  bu[j][i][2][1] =  dt * ty2 * ( u[k][j+1][i][1] * tmp1 );
  bu[j][i][3][1] = 0.0;
  bu[j][i][4][1] = 0.0;

  bu[j][i][0][2] =  dt * ty2
    * ( - ( u[k][j+1][i][2] * tmp1 ) * ( u[k][j+1][i][2] * tmp1 )
        + C2 * ( qs[k][j+1][i] * tmp1 ) )
    - dt * ty1 * ( - r43 * c34 * tmp2 * u[k][j+1][i][2] );
  bu[j][i][1][2] =  dt * ty2
    * ( - C2 * ( u[k][j+1][i][1] * tmp1 ) );
  bu[j][i][2][2] =  dt * ty2 * ( ( 2.0 - C2 )
      * ( u[k][j+1][i][2] * tmp1 ) )
    - dt * ty1 * ( r43 * c34 * tmp1 )
    - dt * ty1 * dy3;
  bu[j][i][3][2] =  dt * ty2
    * ( - C2 * ( u[k][j+1][i][3] * tmp1 ) );
  bu[j][i][4][2] =  dt * ty2 * C2;

  bu[j][i][0][3] =  dt * ty2
    * ( - ( u[k][j+1][i][2]*u[k][j+1][i][3] ) * tmp2 )
    - dt * ty1 * ( - c34 * tmp2 * u[k][j+1][i][3] );
  bu[j][i][1][3] = 0.0;
  bu[j][i][2][3] =  dt * ty2 * ( u[k][j+1][i][3] * tmp1 );
  bu[j][i][3][3] =  dt * ty2 * ( u[k][j+1][i][2] * tmp1 )
    - dt * ty1 * ( c34 * tmp1 )
    - dt * ty1 * dy4;
  bu[j][i][4][3] = 0.0;

  bu[j][i][0][4] =  dt * ty2
    * ( ( C2 * 2.0 * qs[k][j+1][i]
          - C1 * u[k][j+1][i][4] )
        * ( u[k][j+1][i][2] * tmp2 ) )
    - dt * ty1
    * ( - (     c34 - c1345 )*tmp3*(u[k][j+1][i][1]*u[k][j+1][i][1])
        - ( r43*c34 - c1345 )*tmp3*(u[k][j+1][i][2]*u[k][j+1][i][2])
        - (     c34 - c1345 )*tmp3*(u[k][j+1][i][3]*u[k][j+1][i][3])
        - c1345*tmp2*u[k][j+1][i][4] );
  bu[j][i][1][4] =  dt * ty2
    * ( - C2 * ( u[k][j+1][i][1]*u[k][j+1][i][2] ) * tmp2 )
    - dt * ty1
    * ( c34 - c1345 ) * tmp2 * u[k][j+1][i][1];
  bu[j][i][2][4] =  dt * ty2
    * ( C1 * ( u[k][j+1][i][4] * tmp1 )
        - C2 
        * ( qs[k][j+1][i] * tmp1
          + u[k][j+1][i][2]*u[k][j+1][i][2] * tmp2 ) )
    - dt * ty1
    * ( r43*c34 - c1345 ) * tmp2 * u[k][j+1][i][2];
  bu[j][i][3][4] =  dt * ty2
    * ( - C2 * ( u[k][j+1][i][2]*u[k][j+1][i][3] ) * tmp2 )
    - dt * ty1 * ( c34 - c1345 ) * tmp2 * u[k][j+1][i][3];
  bu[j][i][4][4] =  dt * ty2
    * ( C1 * ( u[k][j+1][i][2] * tmp1 ) )
    - dt * ty1 * c1345 * tmp1
    - dt * ty1 * dy5;

  //---------------------------------------------------------------------
  // form the third block sub-diagonal
  //---------------------------------------------------------------------
  tmp1 = rho_i[k+1][j][i];
  tmp2 = tmp1 * tmp1;
  tmp3 = tmp1 * tmp2;

  cu[j][i][0][0] = - dt * tz1 * dz1;
  cu[j][i][1][0] =   0.0;
  cu[j][i][2][0] =   0.0;
  cu[j][i][3][0] = dt * tz2;
  cu[j][i][4][0] =   0.0;

  cu[j][i][0][1] = dt * tz2
    * ( - ( u[k+1][j][i][1]*u[k+1][j][i][3] ) * tmp2 )
    - dt * tz1 * ( - c34 * tmp2 * u[k+1][j][i][1] );
  cu[j][i][1][1] = dt * tz2 * ( u[k+1][j][i][3] * tmp1 )
    - dt * tz1 * c34 * tmp1
    - dt * tz1 * dz2;
  cu[j][i][2][1] = 0.0;
  cu[j][i][3][1] = dt * tz2 * ( u[k+1][j][i][1] * tmp1 );
  cu[j][i][4][1] = 0.0;

  cu[j][i][0][2] = dt * tz2
    * ( - ( u[k+1][j][i][2]*u[k+1][j][i][3] ) * tmp2 )
    - dt * tz1 * ( - c34 * tmp2 * u[k+1][j][i][2] );
  cu[j][i][1][2] = 0.0;
  cu[j][i][2][2] = dt * tz2 * ( u[k+1][j][i][3] * tmp1 )
    - dt * tz1 * ( c34 * tmp1 )
    - dt * tz1 * dz3;
  cu[j][i][3][2] = dt * tz2 * ( u[k+1][j][i][2] * tmp1 );
  cu[j][i][4][2] = 0.0;

  cu[j][i][0][3] = dt * tz2
    * ( - ( u[k+1][j][i][3] * tmp1 ) * ( u[k+1][j][i][3] * tmp1 )
        + C2 * ( qs[k+1][j][i] * tmp1 ) )
    - dt * tz1 * ( - r43 * c34 * tmp2 * u[k+1][j][i][3] );
  cu[j][i][1][3] = dt * tz2
    * ( - C2 * ( u[k+1][j][i][1] * tmp1 ) );
  cu[j][i][2][3] = dt * tz2
    * ( - C2 * ( u[k+1][j][i][2] * tmp1 ) );
  cu[j][i][3][3] = dt * tz2 * ( 2.0 - C2 )
    * ( u[k+1][j][i][3] * tmp1 )
    - dt * tz1 * ( r43 * c34 * tmp1 )
    - dt * tz1 * dz4;
  cu[j][i][4][3] = dt * tz2 * C2;

  cu[j][i][0][4] = dt * tz2
    * ( ( C2 * 2.0 * qs[k+1][j][i]
          - C1 * u[k+1][j][i][4] )
        * ( u[k+1][j][i][3] * tmp2 ) )
    - dt * tz1
    * ( - ( c34 - c1345 ) * tmp3 * (u[k+1][j][i][1]*u[k+1][j][i][1])
        - ( c34 - c1345 ) * tmp3 * (u[k+1][j][i][2]*u[k+1][j][i][2])
        - ( r43*c34 - c1345 )* tmp3 * (u[k+1][j][i][3]*u[k+1][j][i][3])
        - c1345 * tmp2 * u[k+1][j][i][4] );
  cu[j][i][1][4] = dt * tz2
    * ( - C2 * ( u[k+1][j][i][1]*u[k+1][j][i][3] ) * tmp2 )
    - dt * tz1 * ( c34 - c1345 ) * tmp2 * u[k+1][j][i][1];
  cu[j][i][2][4] = dt * tz2
    * ( - C2 * ( u[k+1][j][i][2]*u[k+1][j][i][3] ) * tmp2 )
    - dt * tz1 * ( c34 - c1345 ) * tmp2 * u[k+1][j][i][2];
  cu[j][i][3][4] = dt * tz2
    * ( C1 * ( u[k+1][j][i][4] * tmp1 )
        - C2
        * ( qs[k+1][j][i] * tmp1
          + u[k+1][j][i][3]*u[k+1][j][i][3] * tmp2 ) )
    - dt * tz1 * ( r43*c34 - c1345 ) * tmp2 * u[k+1][j][i][3];
  cu[j][i][4][4] = dt * tz2
    * ( C1 * ( u[k+1][j][i][3] * tmp1 ) )
    - dt * tz1 * c1345 * tmp1
    - dt * tz1 * dz5;


}

__global__
void k_buts_KL_sync(double *m_rsd,
                    double *m_u,
                    double *m_qs,
                    double *m_rho_i,
                    double *m_au,
                    double *m_bu,
                    double *m_cu,
                    double *m_du,
                    int nz, int ny, int nx,
                    int wf_sum, int wf_base_k, int wf_base_j,
                    int jst, int jend, 
                    int ist, int iend, 
                    int temp_kst, int temp_kend)
{
  int k, j, i, m;
  double tmp, tmp1;
  double tmat[5][5], tv[5];
  double (*rsd)[ISIZ2/2*2+1][ISIZ1/2*2+1][5];

  double (*au)[5];
  double (*bu)[5];
  double (*cu)[5];
  double (*du)[5];

  int gid1 = blockDim.y * blockIdx.y + threadIdx.y;
  int gid0 = blockDim.x * blockIdx.x + threadIdx.x;

  k = gid1 + temp_kst + wf_base_k;
  j = gid0 + jst + wf_base_j;
  i = wf_sum - gid1 - gid0 - wf_base_k - wf_base_j + ist;

  if (k >= temp_kend || j >= jend || i < ist || i >= iend) return;

  rsd = (double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5])m_rsd;

  au = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_au)[k][j][i];
  bu = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_bu)[k][j][i];
  cu = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_cu)[k][j][i];
  du = ((double (*)[ISIZ2/2*2+1][ISIZ1/2*2+1][5][5])m_du)[k][j][i];


  for (m = 0; m < 5; m++) {
    tv[m] = 
      omega * (  cu[0][m] * rsd[k+1][j][i][0]
          + cu[1][m] * rsd[k+1][j][i][1]
          + cu[2][m] * rsd[k+1][j][i][2]
          + cu[3][m] * rsd[k+1][j][i][3]
          + cu[4][m] * rsd[k+1][j][i][4] );
  }
  for (m = 0; m < 5; m++) {
    tv[m] = tv[m]
      + omega * ( bu[0][m] * rsd[k][j+1][i][0]
          + au[0][m] * rsd[k][j][i+1][0]
          + bu[1][m] * rsd[k][j+1][i][1]
          + au[1][m] * rsd[k][j][i+1][1]
          + bu[2][m] * rsd[k][j+1][i][2]
          + au[2][m] * rsd[k][j][i+1][2]
          + bu[3][m] * rsd[k][j+1][i][3]
          + au[3][m] * rsd[k][j][i+1][3]
          + bu[4][m] * rsd[k][j+1][i][4]
          + au[4][m] * rsd[k][j][i+1][4] );
  }

  //---------------------------------------------------------------------
  // diagonal block inversion
  //---------------------------------------------------------------------
  for (m = 0; m < 5; m++) {
    tmat[m][0] = du[0][m];
    tmat[m][1] = du[1][m];
    tmat[m][2] = du[2][m];
    tmat[m][3] = du[3][m];
    tmat[m][4] = du[4][m];
  }

  tmp1 = 1.0 / tmat[0][0];
  tmp = tmp1 * tmat[1][0];
  tmat[1][1] =  tmat[1][1] - tmp * tmat[0][1];
  tmat[1][2] =  tmat[1][2] - tmp * tmat[0][2];
  tmat[1][3] =  tmat[1][3] - tmp * tmat[0][3];
  tmat[1][4] =  tmat[1][4] - tmp * tmat[0][4];
  tv[1] = tv[1] - tv[0] * tmp;

  tmp = tmp1 * tmat[2][0];
  tmat[2][1] =  tmat[2][1] - tmp * tmat[0][1];
  tmat[2][2] =  tmat[2][2] - tmp * tmat[0][2];
  tmat[2][3] =  tmat[2][3] - tmp * tmat[0][3];
  tmat[2][4] =  tmat[2][4] - tmp * tmat[0][4];
  tv[2] = tv[2] - tv[0] * tmp;

  tmp = tmp1 * tmat[3][0];
  tmat[3][1] =  tmat[3][1] - tmp * tmat[0][1];
  tmat[3][2] =  tmat[3][2] - tmp * tmat[0][2];
  tmat[3][3] =  tmat[3][3] - tmp * tmat[0][3];
  tmat[3][4] =  tmat[3][4] - tmp * tmat[0][4];
  tv[3] = tv[3] - tv[0] * tmp;

  tmp = tmp1 * tmat[4][0];
  tmat[4][1] =  tmat[4][1] - tmp * tmat[0][1];
  tmat[4][2] =  tmat[4][2] - tmp * tmat[0][2];
  tmat[4][3] =  tmat[4][3] - tmp * tmat[0][3];
  tmat[4][4] =  tmat[4][4] - tmp * tmat[0][4];
  tv[4] = tv[4] - tv[0] * tmp;

  tmp1 = 1.0 / tmat[1][1];
  tmp = tmp1 * tmat[2][1];
  tmat[2][2] =  tmat[2][2] - tmp * tmat[1][2];
  tmat[2][3] =  tmat[2][3] - tmp * tmat[1][3];
  tmat[2][4] =  tmat[2][4] - tmp * tmat[1][4];
  tv[2] = tv[2] - tv[1] * tmp;

  tmp = tmp1 * tmat[3][1];
  tmat[3][2] =  tmat[3][2] - tmp * tmat[1][2];
  tmat[3][3] =  tmat[3][3] - tmp * tmat[1][3];
  tmat[3][4] =  tmat[3][4] - tmp * tmat[1][4];
  tv[3] = tv[3] - tv[1] * tmp;

  tmp = tmp1 * tmat[4][1];
  tmat[4][2] =  tmat[4][2] - tmp * tmat[1][2];
  tmat[4][3] =  tmat[4][3] - tmp * tmat[1][3];
  tmat[4][4] =  tmat[4][4] - tmp * tmat[1][4];
  tv[4] = tv[4] - tv[1] * tmp;

  tmp1 = 1.0 / tmat[2][2];
  tmp = tmp1 * tmat[3][2];
  tmat[3][3] =  tmat[3][3] - tmp * tmat[2][3];
  tmat[3][4] =  tmat[3][4] - tmp * tmat[2][4];
  tv[3] = tv[3] - tv[2] * tmp;

  tmp = tmp1 * tmat[4][2];
  tmat[4][3] =  tmat[4][3] - tmp * tmat[2][3];
  tmat[4][4] =  tmat[4][4] - tmp * tmat[2][4];
  tv[4] = tv[4] - tv[2] * tmp;

  tmp1 = 1.0 / tmat[3][3];
  tmp = tmp1 * tmat[4][3];
  tmat[4][4] =  tmat[4][4] - tmp * tmat[3][4];
  tv[4] = tv[4] - tv[3] * tmp;

  //---------------------------------------------------------------------
  // back substitution
  //---------------------------------------------------------------------
  tv[4] = tv[4] / tmat[4][4];

  tv[3] = tv[3] - tmat[3][4] * tv[4];
  tv[3] = tv[3] / tmat[3][3];

  tv[2] = tv[2]
    - tmat[2][3] * tv[3]
    - tmat[2][4] * tv[4];
  tv[2] = tv[2] / tmat[2][2];

  tv[1] = tv[1]
    - tmat[1][2] * tv[2]
    - tmat[1][3] * tv[3]
    - tmat[1][4] * tv[4];
  tv[1] = tv[1] / tmat[1][1];

  tv[0] = tv[0]
    - tmat[0][1] * tv[1]
    - tmat[0][2] * tv[2]
    - tmat[0][3] * tv[3]
    - tmat[0][4] * tv[4];
  tv[0] = tv[0] / tmat[0][0];

  rsd[k][j][i][0] = rsd[k][j][i][0] - tv[0];
  rsd[k][j][i][1] = rsd[k][j][i][1] - tv[1];
  rsd[k][j][i][2] = rsd[k][j][i][2] - tv[2];
  rsd[k][j][i][3] = rsd[k][j][i][3] - tv[3];
  rsd[k][j][i][4] = rsd[k][j][i][4] - tv[4];



}
