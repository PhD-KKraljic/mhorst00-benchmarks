#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is an CUDA® C version of the NPB LU code. This CUDA® C  //
//  version is a part of SNU-NPB 2019 developed by the Center for Manycore //
//  Programming at Seoul National University and derived from the serial   //
//  Fortran versions in "NPB3.3.1-SER" developed by NAS.                   //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on original NPB 3.3.1, including the technical report, the //
//  original specifications, source code, results and information on how   //
//  to submit new results, is available at:                                //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Information on SNU-NPB 2019, including the conference paper and source //
//  code, is available at:                                                 //
//                                                                         //
//           http://aces.snu.ac.kr                                         //
//                                                                         //
//  Send comments or suggestions for this CUDA® C version to               //
//  snunpb@aces.snu.ac.kr                                                  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 08826, Korea                                             //
//                                                                         //
//          E-mail: snunpb@aces.snu.ac.kr                                  //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Youngdong Do, Hyung Mo Kim, Pyeongseok Oh, Daeyoung Park,      //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include <stdio.h>
#include <math.h>
#include "applu.incl"

extern "C" {
#include "timers.h"
}

//---------------------------------------------------------------------
// compute the lower triangular part of the jacobian matrix
//---------------------------------------------------------------------

//---------------------------------------------------------------------
// 
// compute the regular-sparse, block lower triangular solution:
// 
// v <-- ( L-inv ) * v
// 
//---------------------------------------------------------------------
//---------------------------------------------------------------------
// To improve cache performance, second two dimensions padded by 1 
// for even number sizes only.  Only needed in v.
//---------------------------------------------------------------------

hipEvent_t       (*ev_k_jacld_sync)[2],
                  (*ev_k_blts_BR_sync)[2],
                  (*ev_k_blts_KL_sync)[2],
                  (*ev_k_jbl_datagen_sync)[2],
                  (*ev_k_jbl_datacopy_sync)[2];

static enum PropSyncAlgo   jbl_prop_algo_sync;

static void jacld_sync(int work_step, 
                       int work_num_item);

static hipEvent_t* blts_BR_sync(int work_step, 
                                 int work_num_item);

static hipEvent_t* blts_KL_sync(int work_step, 
                                 int work_num_item);

void jacld_blts_init_sync(int iter, int item_default,
                          int blk_size_k, int blk_size)
{
  int i;
  int tmp_work_base, tmp_work_num_item;
  double start_t, end_t, KL_time, BR_time;

  ev_k_jacld_sync = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_blts_BR_sync = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_blts_KL_sync = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_jbl_datagen_sync = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_jbl_datacopy_sync = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);

  for (i = 0; i < iter; i++) {
    hipEventCreate(&ev_k_jacld_sync[i][0]);
    hipEventCreate(&ev_k_blts_BR_sync[i][0]);
    hipEventCreate(&ev_k_blts_KL_sync[i][0]);
    hipEventCreate(&ev_k_jbl_datagen_sync[i][0]);
    hipEventCreate(&ev_k_jbl_datacopy_sync[i][0]);

    hipEventCreate(&ev_k_jacld_sync[i][1]);
    hipEventCreate(&ev_k_blts_BR_sync[i][1]);
    hipEventCreate(&ev_k_blts_KL_sync[i][1]);
    hipEventCreate(&ev_k_jbl_datagen_sync[i][1]);
    hipEventCreate(&ev_k_jbl_datacopy_sync[i][1]);
  }

  // warm up before profiling
  for (i = 0; i < iter; i++) {
    tmp_work_base = i*item_default + 1;
    tmp_work_num_item = min(item_default, nz-1 - tmp_work_base);

    blts_KL_sync(i, tmp_work_num_item);
    blts_BR_sync(i, tmp_work_num_item);

    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));
  }

  // profiling for KL
  timer_clear(t_jbl_KL_prof);
  start_t = timer_read(t_jbl_KL_prof);
  timer_start(t_jbl_KL_prof);

  for (i = 0; i < iter; i++) {
    tmp_work_base = i*item_default + 1;
    tmp_work_num_item = min(item_default, nz-1 - tmp_work_base);

    blts_KL_sync(i, tmp_work_num_item);

    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));
  }

  timer_stop(t_jbl_KL_prof);
  end_t = timer_read(t_jbl_KL_prof);
  KL_time = end_t - start_t;

  DETAIL_LOG("KL time : %f", KL_time);

  // profiling for BR
  timer_clear(t_jbl_BR_prof);
  start_t = timer_read(t_jbl_BR_prof);
  timer_start(t_jbl_BR_prof);

  for (i = 0; i < iter; i++) {
    tmp_work_base = i*item_default + 1;
    tmp_work_num_item = min(item_default, nz-1 - tmp_work_base);
    blts_BR_sync(i, tmp_work_num_item);

    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));
  }

  timer_stop(t_jbl_BR_prof);
  end_t = timer_read(t_jbl_BR_prof);
  BR_time = end_t - start_t;

  DETAIL_LOG("BR time : %f", BR_time);

  if (KL_time < BR_time)
    jbl_prop_algo_sync = KERNEL_LAUNCH;
  else
    jbl_prop_algo_sync = BARRIER;

  if (jbl_prop_algo_sync == KERNEL_LAUNCH)
    DETAIL_LOG("jacld blts computation policy : Kernel launch");
  else
    DETAIL_LOG("jacld blts computation policy : Kernel launch + barrier");

}

void jacld_blts_release_sync(int iter)
{
  int i;

  for (i = 0; i < iter; i++) {
    hipEventDestroy(ev_k_jacld_sync[i][0]);
    hipEventDestroy(ev_k_blts_BR_sync[i][0]);
    hipEventDestroy(ev_k_blts_KL_sync[i][0]);
    hipEventDestroy(ev_k_jbl_datagen_sync[i][0]);
    hipEventDestroy(ev_k_jbl_datacopy_sync[i][0]);

    hipEventDestroy(ev_k_jacld_sync[i][1]);
    hipEventDestroy(ev_k_blts_BR_sync[i][1]);
    hipEventDestroy(ev_k_blts_KL_sync[i][1]);
    hipEventDestroy(ev_k_jbl_datagen_sync[i][1]);
    hipEventDestroy(ev_k_jbl_datacopy_sync[i][1]);
  }

  free(ev_k_jacld_sync);
  free(ev_k_blts_BR_sync);
  free(ev_k_blts_KL_sync);
  free(ev_k_jbl_datagen_sync);
  free(ev_k_jbl_datacopy_sync);
}


hipEvent_t* jacld_blts_body_sync(int work_step,
                                  int work_max_iter,
                                  int work_base,
                                  int work_num_item)
{
  dim3 numBlocks, numThreads;
  size_t lws[3], gws[3];
  int kend = (int)nz-1;
  hipEvent_t *ev_prop_end;
  int buf_idx = (work_step%2)*buffering_flag;
  int next_buf_idx = ((work_step+1)%2)*buffering_flag;

  // ##################
  //  Kernel Execution
  // ##################

  if(split_flag){

    lws[2] = 1;
    lws[1] = 1;
    lws[0] = max_work_group_size;

    gws[2] = work_num_item + 1;
    gws[1] = jend;
    gws[0] = iend;

    gws[2] = RoundWorkSize(gws[2], lws[2]);
    gws[1] = RoundWorkSize(gws[1], lws[1]);
    gws[0] = RoundWorkSize(gws[0], lws[0]);

    numBlocks.x = gws[0] / lws[0];
    numBlocks.y = gws[1] / lws[1];
    numBlocks.z = gws[2] / lws[2];
    numThreads.x = lws[0];
    numThreads.y = lws[1];
    numThreads.z = lws[2];

    CUCHK(hipEventRecord(ev_k_jbl_datagen_sync[work_step][0], cmd_q[KERNEL_Q]));

    cuda_ProfilerStartEventRecord("k_jbl_datagen_sync",  cmd_q[KERNEL_Q]);
    k_jbl_datagen_sync<<< numBlocks, numThreads, 0, cmd_q[KERNEL_Q]>>>
      (
       m_u[buf_idx],
       m_qs[buf_idx],
       m_rho_i[buf_idx],
       kend, jend, iend, 
       work_base, work_num_item
       );
    CUCHK(hipGetLastError());
    cuda_ProfilerEndEventRecord("k_jbl_datagen_sync",  cmd_q[KERNEL_Q]);

    CUCHK(hipEventRecord(ev_k_jbl_datagen_sync[work_step][1], cmd_q[KERNEL_Q]));

    if (!buffering_flag)
      CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));
  }


  jacld_sync(work_step, work_num_item);

  if (jbl_prop_algo_sync == BARRIER)
    ev_prop_end = blts_BR_sync(work_step, work_num_item);
  else 
    ev_prop_end = blts_KL_sync(work_step, work_num_item);

  if (!buffering_flag && split_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));


  // THIS SHOULD BE enqueue in kernel command queue
  if (work_step < work_max_iter-1) {

    if (split_flag) {

      lws[1] = 1;
      lws[0] = max_work_group_size;

      gws[1] = jend - jst;
      gws[0] = iend - ist;
      gws[0] *= 5;

      gws[1] = RoundWorkSize(gws[1], lws[1]);
      gws[0] = RoundWorkSize(gws[0], lws[0]);

      numBlocks.x = gws[0] / lws[0];
      numBlocks.y = gws[1] / lws[1];
      numBlocks.z = 1;
      numThreads.x = lws[0];
      numThreads.y = lws[1];
      numThreads.z = 1;

      CUCHK(hipEventRecord(ev_k_jbl_datacopy_sync[work_step][0], cmd_q[KERNEL_Q]));

      cuda_ProfilerStartEventRecord("k_jbl_datacopy_sync",  cmd_q[KERNEL_Q]);
      k_jbl_datacopy_sync<<< numBlocks, numThreads, 0, cmd_q[KERNEL_Q]>>>
        (
         m_rsd[buf_idx], 
         m_rsd[next_buf_idx],
         m_u[buf_idx], 
         m_u[next_buf_idx],
         jst, jend, 
         ist, iend, 
         work_num_item
        );
      CUCHK(hipGetLastError());
      cuda_ProfilerEndEventRecord("k_jbl_datacopy_sync",  cmd_q[KERNEL_Q]);

      CUCHK(hipEventRecord(ev_k_jbl_datacopy_sync[work_step][1], cmd_q[KERNEL_Q]));

      if (!buffering_flag)
        CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));
    }
  }


  if (work_step == work_max_iter - 1)
    return ev_prop_end;
  else
    return &ev_k_jbl_datacopy_sync[work_step][1];

}



static void jacld_sync(int work_step, int work_num_item)
{

  int temp_kend, temp_kst;
  int buf_idx = (work_step%2)*buffering_flag;
  temp_kst = 1; 
  temp_kend = work_num_item + 1;
  size_t lws[3], gws[3];
  dim3 numBlocks, numThreads;

  gws[2] = work_num_item;
  gws[1] = jend - jst;
  gws[0] = iend - ist;

  lws[2] = 1;
  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  numBlocks.x = gws[0] / lws[0];
  numBlocks.y = gws[1] / lws[1];
  numBlocks.z = gws[2] / lws[2];
  numThreads.x = lws[0];
  numThreads.y = lws[1];
  numThreads.z = lws[2];

  CUCHK(hipEventRecord(ev_k_jacld_sync[work_step][0], cmd_q[KERNEL_Q]));

  cuda_ProfilerStartEventRecord("k_jacld_sync",  cmd_q[KERNEL_Q]);
  k_jacld_sync<<< numBlocks, numThreads, 0, cmd_q[KERNEL_Q]>>>
    (
     m_rsd[buf_idx], m_u[buf_idx],
     m_qs[buf_idx], m_rho_i[buf_idx],
     m_a, m_b, m_c, m_d,
     nz, ny, nx,
     jst, jend,
     ist, iend,
     temp_kst, temp_kend
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_jacld_sync",  cmd_q[KERNEL_Q]);

  CUCHK(hipEventRecord(ev_k_jacld_sync[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

}


static hipEvent_t* blts_BR_sync(int work_step, int work_num_item)
{
  size_t lws[3], gws[3];
  int temp_kst, temp_kend,
      num_k, num_j, num_i,
      num_block_k, num_block_j, num_block_i,
      wg00_block_k, wg00_block_j, wg00_block_i,
      wg00_head_k, wg00_head_j, wg00_head_i,
      depth, diagonals,
      num_wg,
      iter, max_iter;

  int buf_idx = (work_step%2)*buffering_flag;

  temp_kst = 1; temp_kend = work_num_item + 1;

  num_k = temp_kend - temp_kst; 
  num_j = jend - jst;
  num_i = iend - ist;

  num_block_k = ( ( num_k - 1) / block_size_k ) + 1;
  num_block_j = ( ( num_j - 1) / block_size ) + 1;
  num_block_i = ( ( num_i - 1) / block_size ) + 1;

  wg00_head_k = temp_kst;
  wg00_head_j = jst;
  wg00_head_i = ist;


  max_iter = num_block_k + num_block_j + num_block_i - 2;

  CUCHK(hipEventRecord(ev_k_blts_BR_sync[work_step][0], cmd_q[KERNEL_Q]));

  // blocking iteration
  for (iter = 0; iter < max_iter ; iter++) {

    num_wg = 0;

    wg00_block_k = (wg00_head_k-temp_kst) / block_size_k;
    wg00_block_j = (wg00_head_j-jst) / block_size;
    wg00_block_i = (wg00_head_i-ist) / block_size;


    // calculate the number of work-group
    // diagonals = the number of active diagonals
    diagonals  = min(wg00_block_k+1, num_block_j+num_block_i-1 - (wg00_block_j+wg00_block_i));

    for(depth = 0; depth < diagonals; depth++){
      // addition factor is the number of blocks in current diagonal
      num_wg += min(wg00_block_j+1, num_block_i - wg00_block_i);

      wg00_block_j++;
      if(wg00_block_j >= num_block_j){
        wg00_block_j--;
        wg00_block_i++;
      }

    }

    // reset the current block position 
    wg00_block_k = (wg00_head_k-temp_kst) / block_size_k;
    wg00_block_j = (wg00_head_j-jst) / block_size;
    wg00_block_i = (wg00_head_i-ist) / block_size;


    lws[0] = jacld_blts_lws;
    gws[0] = lws[0]*num_wg;
    gws[0] = RoundWorkSize(gws[0], lws[0]);

    cuda_ProfilerStartEventRecord("k_blts_BR_sync",  cmd_q[KERNEL_Q]);
    k_blts_BR_sync<<< gws[0] / lws[0], lws[0], 0, cmd_q[KERNEL_Q]>>>
      (
       m_rsd[buf_idx], m_rho_i[buf_idx],
       m_u[buf_idx], m_qs[buf_idx],
       m_a, m_b, m_c, m_d,
       temp_kst, temp_kend,
       jst, jend,
       ist, iend,
       wg00_head_k, wg00_head_j, wg00_head_i,
       wg00_block_k, wg00_block_j, wg00_block_i,
       num_block_k, num_block_j, num_block_i,
       block_size, block_size_k
      );
    CUCHK(hipGetLastError());
    cuda_ProfilerEndEventRecord("k_blts_BR_sync",  cmd_q[KERNEL_Q]);

    wg00_head_k += block_size_k;
    if (wg00_head_k >= temp_kend) {
      wg00_head_k -= block_size_k;
      wg00_head_j += block_size;
      if (wg00_head_j >= jend) {
        wg00_head_j -= block_size;
        wg00_head_i += block_size;
      }
    }

  }

  CUCHK(hipEventRecord(ev_k_blts_BR_sync[work_step][1], cmd_q[KERNEL_Q]));

  return &ev_k_blts_BR_sync[work_step][1];
}

static hipEvent_t* blts_KL_sync(int work_step, int work_num_item)
{

  int k;
  int temp;
  int temp_kend, temp_kst;
  int buf_idx = (work_step%2)*buffering_flag;
  temp_kst = 1; 
  temp_kend = work_num_item + 1;
  dim3 numBlocks, numThreads;
  size_t lws[3], gws[3];
  int lbk, ubk, lbj, ubj;

  size_t blts_max_work_group_size = 64;
  size_t blts_max_work_items_sizes = 64;

  CUCHK(hipEventRecord(ev_k_blts_KL_sync[work_step][0], cmd_q[KERNEL_Q]));

  for (k = 0; k <= (temp_kend-temp_kst-1)+(iend-ist-1)+(jend-jst-1); k++) {
    lbk = (k-(iend-ist-1)-(jend-jst-1)) >= 0 ? (k-(iend-ist-1)-(jend-jst-1)) : 0;
    ubk = k < (temp_kend-temp_kst-1) ? k : (temp_kend-temp_kst-1);
    lbj = (k-(iend-ist-1)-(temp_kend-temp_kst)) >= 0 ? (k-(iend-ist-1)-(temp_kend-temp_kst)) : 0;
    ubj = k < (jend-jst-1) ? k : (jend-jst-1);

    lws[0] = (ubj-lbj+1) < (int)blts_max_work_items_sizes? (ubj-lbj+1) : blts_max_work_items_sizes;
    temp = blts_max_work_group_size / lws[0];
    lws[1] = (ubk-lbk+1) < temp ? (ubk-lbk+1) : temp;
    lws[2] = 1;

    gws[2] = 1;
    gws[1] = (size_t)(ubk-lbk+1);
    gws[0] = (size_t)(ubj-lbj+1);

    gws[2] = RoundWorkSize(gws[2], lws[2]);
    gws[1] = RoundWorkSize(gws[1], lws[1]);
    gws[0] = RoundWorkSize(gws[0], lws[0]);

    numBlocks.x = gws[0] / lws[0];
    numBlocks.y = gws[1] / lws[1];
    numBlocks.z = gws[2] / lws[2];
    numThreads.x = lws[0];
    numThreads.y = lws[1];
    numThreads.z = lws[2];

    cuda_ProfilerStartEventRecord("k_blts_KL_baseline",  cmd_q[KERNEL_Q]);
    k_blts_KL_baseline<<< numBlocks, numThreads, 0, cmd_q[KERNEL_Q]>>>
      (
       m_rsd[buf_idx], m_u[buf_idx],
       m_qs[buf_idx], m_rho_i[buf_idx],
       m_a, m_b, m_c, m_d,
       nz, ny, nx,
       k, lbk,
       lbj, jst, jend,
       ist, iend,
       temp_kst, temp_kend
      );
    CUCHK(hipGetLastError());
    cuda_ProfilerEndEventRecord("k_blts_KL_baseline",  cmd_q[KERNEL_Q]);
  }

  CUCHK(hipEventRecord(ev_k_blts_KL_sync[work_step][1], cmd_q[KERNEL_Q]));

  return &ev_k_blts_KL_sync[work_step][1];
}
