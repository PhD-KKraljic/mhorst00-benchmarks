#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is an CUDA® C version of the NPB LU code. This CUDA® C  //
//  version is a part of SNU-NPB 2019 developed by the Center for Manycore //
//  Programming at Seoul National University and derived from the serial   //
//  Fortran versions in "NPB3.3.1-SER" developed by NAS.                   //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on original NPB 3.3.1, including the technical report, the //
//  original specifications, source code, results and information on how   //
//  to submit new results, is available at:                                //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Information on SNU-NPB 2019, including the conference paper and source //
//  code, is available at:                                                 //
//                                                                         //
//           http://aces.snu.ac.kr                                         //
//                                                                         //
//  Send comments or suggestions for this CUDA® C version to               //
//  snunpb@aces.snu.ac.kr                                                  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 08826, Korea                                             //
//                                                                         //
//          E-mail: snunpb@aces.snu.ac.kr                                  //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Youngdong Do, Hyung Mo Kim, Pyeongseok Oh, Daeyoung Park,      //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include <math.h>
#include <stdio.h>
#include <assert.h>

#include "applu.incl"
extern "C" {
#include "timers.h"
}

hipEvent_t       (*ev_k_rhs1_gmem)[2],
                  (*ev_k_rhs1_datagen_gmem)[2],
                  (*ev_k_rhsx_gmem)[2],
                  (*ev_k_rhsy_gmem)[2],
                  (*ev_k_rhsz_gmem)[2];

void rhs_init_gmem(int iter)
{
  int i;
  ev_k_rhs1_gmem = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhs1_datagen_gmem = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhsx_gmem = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhsy_gmem = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhsz_gmem = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);

  for (i = 0; i < iter; i++) {
    hipEventCreate(&ev_k_rhs1_gmem[i][0]);
    hipEventCreate(&ev_k_rhs1_datagen_gmem[i][0]);
    hipEventCreate(&ev_k_rhsx_gmem[i][0]);
    hipEventCreate(&ev_k_rhsy_gmem[i][0]);
    hipEventCreate(&ev_k_rhsz_gmem[i][0]);

    hipEventCreate(&ev_k_rhs1_gmem[i][1]);
    hipEventCreate(&ev_k_rhs1_datagen_gmem[i][1]);
    hipEventCreate(&ev_k_rhsx_gmem[i][1]);
    hipEventCreate(&ev_k_rhsy_gmem[i][1]);
    hipEventCreate(&ev_k_rhsz_gmem[i][1]);
  }
}

void rhs_release_gmem(int iter)
{
  int i;

  for (i = 0; i < iter; i++) {
    hipEventDestroy(ev_k_rhs1_gmem[i][0]);
    hipEventDestroy(ev_k_rhs1_datagen_gmem[i][0]);
    hipEventDestroy(ev_k_rhsx_gmem[i][0]);
    hipEventDestroy(ev_k_rhsy_gmem[i][0]);
    hipEventDestroy(ev_k_rhsz_gmem[i][0]);

    hipEventDestroy(ev_k_rhs1_gmem[i][1]);
    hipEventDestroy(ev_k_rhs1_datagen_gmem[i][1]);
    hipEventDestroy(ev_k_rhsx_gmem[i][1]);
    hipEventDestroy(ev_k_rhsy_gmem[i][1]);
    hipEventDestroy(ev_k_rhsz_gmem[i][1]);
  }

  free(ev_k_rhs1_gmem);
  free(ev_k_rhs1_datagen_gmem);
  free(ev_k_rhsx_gmem);
  free(ev_k_rhsy_gmem);
  free(ev_k_rhsz_gmem);
}

hipEvent_t* rhs_body_gmem(int work_step, 
                           int work_base, 
                           int work_num_item, 
                           int copy_buffer_base, 
                           int copy_num_item, 
                           hipEvent_t* ev_wb_ptr)
{
  size_t lws[3], gws[3];
  dim3 numBlocks, numThreads;

  int buf_idx = (work_step%2)*buffering_flag;

  if (timeron) timer_start(t_rhs);

  // ################
  // kernel execution
  // ################
  lws[2] = 1;
  lws[1] = 1;
  lws[0] = min(max_work_item_sizes[0], (int)max_work_group_size);

  gws[2] = (size_t) work_num_item;
  gws[1] = (size_t) ny;
  gws[0] = (size_t) nx*5;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  numBlocks.x = gws[0] / lws[0];
  numBlocks.y = gws[1] / lws[1];
  numBlocks.z = gws[2] / lws[2];
  numThreads.x = lws[0];
  numThreads.y = lws[1];
  numThreads.z = lws[2];

  if (buffering_flag)
    CUCHK(hipStreamWaitEvent(cmd_q[KERNEL_Q], *ev_wb_ptr, 0));

  CUCHK(hipEventRecord(ev_k_rhs1_gmem[work_step][0], cmd_q[KERNEL_Q]));

  cuda_ProfilerStartEventRecord("k_rhs1_gmem",  cmd_q[KERNEL_Q]);
  k_rhs1_gmem<<<numBlocks, numThreads, 0, cmd_q[KERNEL_Q]>>>
    (
     m_rsd[buf_idx], m_frct[buf_idx],
     nx, ny, nz,
     work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_rhs1_gmem",  cmd_q[KERNEL_Q]);

  CUCHK(hipEventRecord(ev_k_rhs1_gmem[work_step][1], cmd_q[KERNEL_Q]));

  if (split_flag && !buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  lws[2] = 1;
  lws[1] = 1;
  lws[0] = min(max_work_item_sizes[0], (int)max_work_group_size);

  gws[2] = (size_t) copy_num_item;
  gws[1] = (size_t) ny;
  gws[0] = (size_t) nx;
 
  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  numBlocks.x = gws[0] / lws[0];
  numBlocks.y = gws[1] / lws[1];
  numBlocks.z = gws[2] / lws[2];
  numThreads.x = lws[0];
  numThreads.y = lws[1];
  numThreads.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhs1_datagen_gmem[work_step][0], cmd_q[KERNEL_Q]));

  cuda_ProfilerStartEventRecord("k_rhs1_datagen_gmem",  cmd_q[KERNEL_Q]);
  k_rhs1_datagen_gmem<<<numBlocks, numThreads, 0, cmd_q[KERNEL_Q]>>>
    (
     m_u[buf_idx], m_rho_i[buf_idx], m_qs[buf_idx],
     nx, ny, nz,
     copy_buffer_base, copy_num_item
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_rhs1_datagen_gmem",  cmd_q[KERNEL_Q]);

  CUCHK(hipEventRecord(ev_k_rhs1_datagen_gmem[work_step][1], cmd_q[KERNEL_Q]));

  if (split_flag && !buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  if (timeron) timer_start(t_rhsx);

  lws[2] = 1;
  lws[1] = 1;
  lws[0] = min(jend-jst, (int)max_work_group_size);
  lws[0] = min((int)lws[0], max_work_item_sizes[0]);

  gws[2] = 1;
  gws[1] = work_num_item;
  gws[0] = jend - jst;
 
  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  numBlocks.x = gws[0] / lws[0];
  numBlocks.y = gws[1] / lws[1];
  numBlocks.z = gws[2] / lws[2];
  numThreads.x = lws[0];
  numThreads.y = lws[1];
  numThreads.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhsx_gmem[work_step][0], cmd_q[KERNEL_Q]));

  cuda_ProfilerStartEventRecord("k_rhsx_gmem",  cmd_q[KERNEL_Q]);
  k_rhsx_gmem<<<numBlocks, numThreads, 0, cmd_q[KERNEL_Q]>>>
    (
     m_u[buf_idx], m_rho_i[buf_idx],
     m_qs[buf_idx], m_rsd[buf_idx],
     jst, jend,
     ist, iend,
     tx1, tx2, tx3,
     dx1, dx2, dx3, dx4, dx5,
     dssp, nx, nz,
     work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_rhsx_gmem",  cmd_q[KERNEL_Q]);

  CUCHK(hipEventRecord(ev_k_rhsx_gmem[work_step][1], cmd_q[KERNEL_Q]));

  if (split_flag && !buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  if (timeron) timer_stop(t_rhsx);


  if (timeron) timer_start(t_rhsy);
  //---------------------------------------------------------------------
  // eta-direction flux differences
  //---------------------------------------------------------------------
  lws[2] = 1;
  lws[1] = 1;
  lws[0] = min(iend - ist, max_work_item_sizes[0]);
  lws[0] = min(lws[0], max_work_group_size);

  gws[2] = 1;
  gws[1] = (size_t) work_num_item;
  gws[0] = (size_t) (iend - ist);
 
  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  numBlocks.x = gws[0] / lws[0];
  numBlocks.y = gws[1] / lws[1];
  numBlocks.z = gws[2] / lws[2];
  numThreads.x = lws[0];
  numThreads.y = lws[1];
  numThreads.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhsy_gmem[work_step][0], cmd_q[KERNEL_Q]));

  cuda_ProfilerStartEventRecord("k_rhsy_gmem",  cmd_q[KERNEL_Q]);
  k_rhsy_gmem<<<numBlocks, numThreads, 0, cmd_q[KERNEL_Q]>>>
    (
     m_u[buf_idx], m_rho_i[buf_idx],
     m_qs[buf_idx], m_rsd[buf_idx],
     ist, iend,
     ty1, ty2, ty3,
     dy1, dy2, dy3, dy4, dy5,
     dssp, ny, nz,
     work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_rhsy_gmem",  cmd_q[KERNEL_Q]);

  CUCHK(hipEventRecord(ev_k_rhsy_gmem[work_step][1], cmd_q[KERNEL_Q]));

  if (split_flag && !buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  if (timeron) timer_stop(t_rhsy);



  if (timeron) timer_start(t_rhsz);
  //---------------------------------------------------------------------
  // zeta-direction flux differences
  //---------------------------------------------------------------------
  lws[2] = 1;
  lws[1] = 1;
  lws[0] = min(max_work_item_sizes[0], iend-ist);
  lws[0] = min(lws[0], max_work_group_size);

  gws[2] = 1;
  gws[1] = (size_t) (jend - jst);
  gws[0] = (size_t) (iend - ist);

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  numBlocks.x = gws[0] / lws[0];
  numBlocks.y = gws[1] / lws[1];
  numBlocks.z = gws[2] / lws[2];
  numThreads.x = lws[0];
  numThreads.y = lws[1];
  numThreads.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhsz_gmem[work_step][0], cmd_q[KERNEL_Q]));

  cuda_ProfilerStartEventRecord("k_rhsz_gmem",  cmd_q[KERNEL_Q]);
  k_rhsz_gmem<<<numBlocks, numThreads, 0, cmd_q[KERNEL_Q]>>>
    (
     m_u[buf_idx], m_rho_i[buf_idx],
     m_qs[buf_idx], m_rsd[buf_idx],
     jst, jend,
     ist, iend,
     tz1, tz2, tz3,
     dz1, dz2, dz3, dz4, dz5,
     dssp, nz,
     work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_rhsz_gmem",  cmd_q[KERNEL_Q]);

  CUCHK(hipEventRecord(ev_k_rhsz_gmem[work_step][1], cmd_q[KERNEL_Q]));

  if (split_flag && !buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));
  
  if (timeron) timer_stop(t_rhsz);

  if (timeron) timer_stop(t_rhs);

  return &ev_k_rhsz_gmem[work_step][1];

}
