//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is an CUDA® C version of the NPB BT code. This CUDA® C  //
//  version is a part of SNU-NPB 2019 developed by the Center for Manycore //
//  Programming at Seoul National University and derived from the serial   //
//  Fortran versions in "NPB3.3.1-SER" developed by NAS.                   //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on original NPB 3.3.1, including the technical report, the //
//  original specifications, source code, results and information on how   //
//  to submit new results, is available at:                                //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Information on SNU-NPB 2019, including the conference paper and source //
//  code, is available at:                                                 //
//                                                                         //
//           http://aces.snu.ac.kr                                         //
//                                                                         //
//  Send comments or suggestions for this CUDA® C version to               //
//  snunpb@aces.snu.ac.kr                                                  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 08826, Korea                                             //
//                                                                         //
//          E-mail: snunpb@aces.snu.ac.kr                                  //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Youngdong Do, Hyung Mo Kim, Pyeongseok Oh, Daeyoung Park,      //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include "header.h"

//---------------------------------------------------------------------
// this function returns the exact solution at point xi, eta, zeta  
//---------------------------------------------------------------------
void exact_solution(double xi, double eta, double zeta, double dtemp[5])
{
  int m;

  for (m = 0; m < 5; m++) {
    dtemp[m] =  ce[m][0] +
      xi*(ce[m][1] + xi*(ce[m][4] + xi*(ce[m][7] + xi*ce[m][10]))) +
      eta*(ce[m][2] + eta*(ce[m][5] + eta*(ce[m][8] + eta*ce[m][11])))+
      zeta*(ce[m][3] + zeta*(ce[m][6] + zeta*(ce[m][9] + 
      zeta*ce[m][12])));
  }
}
