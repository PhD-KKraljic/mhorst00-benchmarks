#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is an CUDA® C version of the NPB BT code. This CUDA® C  //
//  version is a part of SNU-NPB 2019 developed by the Center for Manycore //
//  Programming at Seoul National University and derived from the serial   //
//  Fortran versions in "NPB3.3.1-SER" developed by NAS.                   //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on original NPB 3.3.1, including the technical report, the //
//  original specifications, source code, results and information on how   //
//  to submit new results, is available at:                                //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Information on SNU-NPB 2019, including the conference paper and source //
//  code, is available at:                                                 //
//                                                                         //
//           http://aces.snu.ac.kr                                         //
//                                                                         //
//  Send comments or suggestions for this CUDA® C version to               //
//  snunpb@aces.snu.ac.kr                                                  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 08826, Korea                                             //
//                                                                         //
//          E-mail: snunpb@aces.snu.ac.kr                                  //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Youngdong Do, Hyung Mo Kim, Pyeongseok Oh, Daeyoung Park,      //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include <math.h>
#include <stdio.h>
#include <assert.h>
#include "header.h"
#include "cuda_util.h"

void compute_rhs_init_baseline(int iter);
void compute_rhs_release_baseline(int iter);
void compute_rhs_body_baseline(int work_step,
                               int work_base,
                               int work_num_item,
                               int copy_buffer_base,
                               int copy_num_item,
                               int buf_idx,
                               hipEvent_t *ev_wb_end_ptr);

void compute_rhs_init_parallel(int iter);
void compute_rhs_release_parallel(int iter);
void compute_rhs_body_parallel(int work_step,
                               int work_base,
                               int work_num_item,
                               int copy_buffer_base,
                               int copy_num_item,
                               int buf_idx,
                               hipEvent_t *ev_wb_end_ptr);

void compute_rhs_init_baseline(int iter);
void compute_rhs_release_baseline(int iter);
void compute_rhs_body_baseline(int work_step,
                               int work_base,
                               int work_num_item,
                               int copy_buffer_base,
                               int copy_num_item,
                               int buf_idx,
                               hipEvent_t *ev_wb_end_ptr);

void compute_rhs_init(int iter)
{
  switch (g_opt_level) {
    case OPT_BASELINE:
      compute_rhs_init_baseline(iter);
      break;
    case OPT_PARALLEL:
    case OPT_FULL:
      compute_rhs_init_parallel(iter);
      break;
    default :
      compute_rhs_init_baseline(iter);
      break;
  }
}

void compute_rhs_release(int iter)
{
  switch (g_opt_level) {
    case OPT_BASELINE:
      compute_rhs_release_baseline(iter);
      break;
    case OPT_PARALLEL:
    case OPT_FULL:
      compute_rhs_release_parallel(iter);
      break;
    default :
      compute_rhs_release_baseline(iter);
      break;
  }
}

void compute_rhs_body(int work_step, 
                      int work_base, 
                      int work_num_item, 
                      int copy_buffer_base, 
                      int copy_num_item, 
                      int buf_idx,
                      hipEvent_t *ev_wb_end_ptr)
{
  switch (g_opt_level) {
    case OPT_BASELINE:
      compute_rhs_body_baseline(work_step, 
                                work_base, 
                                work_num_item,
                                copy_buffer_base, 
                                copy_num_item,
                                buf_idx, 
                                ev_wb_end_ptr);
      break;
    case OPT_PARALLEL:
    case OPT_FULL:
      compute_rhs_body_parallel(work_step, 
                                work_base, 
                                work_num_item,
                                copy_buffer_base, 
                                copy_num_item,
                                buf_idx, 
                                ev_wb_end_ptr);
      break;
    default :
      compute_rhs_body_baseline(work_step, 
                                work_base, 
                                work_num_item,
                                copy_buffer_base, 
                                copy_num_item,
                                buf_idx, 
                                ev_wb_end_ptr);
      break;
  }
}

void compute_rhs()
{
  int work_step, work_max_iter, work_base;
  int work_num_item;
  int temp_work_num_item_default;
  int copy_buffer_base, copy_num_item, copy_host_base;

  if (timeron) timer_start(t_rhs);

  //---------------------------------------------------------------------
  // compute the reciprocal of density, and the kinetic energy, 
  // and the speed of sound.
  //---------------------------------------------------------------------

  temp_work_num_item_default = (split_flag) ? (work_num_item_default-4) : work_num_item_default;

  // the number of whole items to be processed
  work_max_iter = ( grid_points[2] - 1 ) / temp_work_num_item_default + 1;

  for (work_step = 0; work_step < work_max_iter; work_step++) {

    work_base = work_step*temp_work_num_item_default;

    // the end index of items + 1
    work_num_item = min(temp_work_num_item_default, grid_points[2] - work_base);

    copy_num_item = get_loop1_copy_num_item(work_base, work_num_item);
    copy_buffer_base = get_loop1_copy_buffer_base(work_base);
    copy_host_base = get_loop1_copy_host_base(work_base);

    if (split_flag) {
      CUCHK(hipMemcpyAsync(((unsigned char*)m_u[0]) + sizeof(double)*copy_buffer_base*(JMAXP+1)*(IMAXP+1)*5,
            &(u[copy_host_base][0][0][0]),
            sizeof(double)*copy_num_item*(JMAXP+1)*(IMAXP+1)*5,
            hipMemcpyHostToDevice, cmd_q[KERNEL_Q]));

      CUCHK(hipMemcpyAsync(((unsigned char*)m_forcing[0]) + sizeof(double)*copy_buffer_base*(JMAXP+1)*(IMAXP+1)*5,
            &(forcing[copy_host_base][0][0][0]),
            sizeof(double)*copy_num_item*(JMAXP+1)*(IMAXP+1)*5,
            hipMemcpyHostToDevice, cmd_q[KERNEL_Q]));

      CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));
    }

    compute_rhs_body(work_step, 
                     work_base, 
                     work_num_item,
                     copy_buffer_base, 
                     copy_num_item,
                     0, // buf_idx
                     NULL);

    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

    if (split_flag) {
      CUCHK(hipMemcpyAsync(&(rhs[work_base][0][0][0]),
      ((unsigned char*) m_rhs[0]) + sizeof(double)*2*(JMAXP+1)*(IMAXP+1)*5,
          sizeof(double)*work_num_item*(JMAXP+1)*(IMAXP+1)*5,
          hipMemcpyDeviceToHost, cmd_q[KERNEL_Q]));

      CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));
    }
  }

  if (timeron) timer_stop(t_rhs);
}


