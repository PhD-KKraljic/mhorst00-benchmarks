#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is an CUDA® C version of the NPB BT code. This CUDA® C  //
//  version is a part of SNU-NPB 2019 developed by the Center for Manycore //
//  Programming at Seoul National University and derived from the serial   //
//  Fortran versions in "NPB3.3.1-SER" developed by NAS.                   //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on original NPB 3.3.1, including the technical report, the //
//  original specifications, source code, results and information on how   //
//  to submit new results, is available at:                                //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Information on SNU-NPB 2019, including the conference paper and source //
//  code, is available at:                                                 //
//                                                                         //
//           http://aces.snu.ac.kr                                         //
//                                                                         //
//  Send comments or suggestions for this CUDA® C version to               //
//  snunpb@aces.snu.ac.kr                                                  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 08826, Korea                                             //
//                                                                         //
//          E-mail: snunpb@aces.snu.ac.kr                                  //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Youngdong Do, Hyung Mo Kim, Pyeongseok Oh, Daeyoung Park,      //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include <math.h>
#include <stdio.h>
#include <assert.h>
#include "header.h"

//---------------------------------------------------------------------
// Performs line solves in Z direction by first factoring
// the block-tridiagonal matrix into an upper triangular matrix, 
// and then performing back substitution to solve for the unknow
// vectors of each line.  
// 
// Make sure we treat elements zero to cell_size in the direction
// of the sweep.
//---------------------------------------------------------------------

hipEvent_t   (*ev_k_z_solve_data_gen_fullopt)[2],
              (*ev_k_z_solve1_fullopt)[2],
              (*ev_k_z_solve2_fullopt)[2],
              (*ev_k_z_solve3_fullopt)[2];

void z_solve_init_fullopt(int iter)
{
  int i;

  ev_k_z_solve_data_gen_fullopt = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_z_solve1_fullopt = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_z_solve2_fullopt = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_z_solve3_fullopt = (hipEvent_t (*)[2])malloc(sizeof(hipEvent_t)*2*iter);

  for (i = 0; i < iter; i++) {
    hipEventCreate(&ev_k_z_solve_data_gen_fullopt[i][0]);
    hipEventCreate(&ev_k_z_solve1_fullopt[i][0]);
    hipEventCreate(&ev_k_z_solve2_fullopt[i][0]);
    hipEventCreate(&ev_k_z_solve3_fullopt[i][0]);

    hipEventCreate(&ev_k_z_solve_data_gen_fullopt[i][1]);
    hipEventCreate(&ev_k_z_solve1_fullopt[i][1]);
    hipEventCreate(&ev_k_z_solve2_fullopt[i][1]);
    hipEventCreate(&ev_k_z_solve3_fullopt[i][1]);
  }
}

void z_solve_release_fullopt(int iter)
{
  int i;

  for (i = 0; i < iter; i++) {
    hipEventDestroy(ev_k_z_solve_data_gen_fullopt[i][0]);
    hipEventDestroy(ev_k_z_solve1_fullopt[i][0]);
    hipEventDestroy(ev_k_z_solve2_fullopt[i][0]);
    hipEventDestroy(ev_k_z_solve3_fullopt[i][0]);

    hipEventDestroy(ev_k_z_solve_data_gen_fullopt[i][1]);
    hipEventDestroy(ev_k_z_solve1_fullopt[i][1]);
    hipEventDestroy(ev_k_z_solve2_fullopt[i][1]);
    hipEventDestroy(ev_k_z_solve3_fullopt[i][1]);
  }

  free(ev_k_z_solve_data_gen_fullopt);
  free(ev_k_z_solve1_fullopt);
  free(ev_k_z_solve2_fullopt);
  free(ev_k_z_solve3_fullopt);
}

void z_solve_fullopt(int work_step, 
                     int work_base, 
                     int work_num_item, 
                     int buf_idx,
                     hipEvent_t* ev_wb_end_ptr) 
{
  size_t lws[3];
  size_t gws[3];

  if (timeron) timer_start(t_zsolve);


  if (split_flag) {
    lws[2] = 1;
    lws[1] = 1;
    lws[0] = max_work_group_size;

    gws[2] = (size_t) grid_points[2];
    gws[1] = (size_t) work_num_item;
    gws[0] = (size_t) grid_points[0]-2;

    gws[2] = RoundWorkSize(gws[2], lws[2]);
    gws[1] = RoundWorkSize(gws[1], lws[1]);
    gws[0] = RoundWorkSize(gws[0], lws[0]);

    dim3 blockSize(gws[0]/lws[0], gws[1]/lws[1], gws[2]/lws[2]);
    dim3 threadSize(lws[0], lws[1], lws[2]);

    if (buffering_flag)
      CUCHK(hipStreamWaitEvent(cmd_q[KERNEL_Q], *ev_wb_end_ptr, 0));

    CUCHK(hipEventRecord(ev_k_z_solve_data_gen_fullopt[work_step][0], cmd_q[KERNEL_Q]));
    cuda_ProfilerStartEventRecord("k_z_solve_data_gen_fullopt",  cmd_q[KERNEL_Q]);
    k_z_solve_data_gen_fullopt<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
      (
       m_u[buf_idx], m_square, m_qs,
       grid_points[0], grid_points[1], grid_points[2],
       work_base, work_num_item, split_flag, work_num_item_default_j
      );
    CUCHK(hipGetLastError());
    cuda_ProfilerEndEventRecord("k_z_solve_data_gen_fullopt",  cmd_q[KERNEL_Q]);
    CUCHK(hipEventRecord(ev_k_z_solve_data_gen_fullopt[work_step][1], cmd_q[KERNEL_Q]));

    if (!buffering_flag)
      CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));
  }


  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[1] = work_num_item * 25;
  gws[0] = grid_points[0]-2;

  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  dim3 blockSize(gws[0]/lws[0], gws[1]/lws[1]);
  dim3 threadSize(lws[0], lws[1]);

  CUCHK(hipEventRecord(ev_k_z_solve1_fullopt[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_z_solve1_fullopt",  cmd_q[KERNEL_Q]);
  k_z_solve1_fullopt<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_lhsA, m_lhsB, m_lhsC,
     grid_points[0], grid_points[1], grid_points[2],
     work_base, work_num_item, split_flag, work_num_item_default_j
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_z_solve1_fullopt",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_z_solve1_fullopt[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));


  lws[2] = 1;
  lws[1] = 1;
  lws[0] = min(max_work_group_size, max_work_item_sizes[0]);

  gws[2] = (size_t) grid_points[2]-2;
  gws[1] = (size_t) work_num_item;
  gws[0] = (size_t) grid_points[0]-2;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = gws[2]/lws[2]; threadSize.z = lws[2];

  CUCHK(hipEventRecord(ev_k_z_solve2_fullopt[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_z_solve2_fullopt",  cmd_q[KERNEL_Q]);
  k_z_solve2_fullopt<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_qs, m_square, m_u[buf_idx], m_lhsA, m_lhsB, m_lhsC,
     grid_points[0], grid_points[1], grid_points[2], dz1, dz2, dz3, dz4, dz5,
     c1, c2, c3, c4, tz1, tz2, con43, c3c4, c1345, dt,
     work_base, work_num_item, split_flag, work_num_item_default_j
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_z_solve2_fullopt",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_z_solve2_fullopt[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));


  size_t max_lws_i = min(max_work_group_size/5, local_mem_size/(sizeof(double)*(3*5*5+2*5)));

  lws[1] = 5;
  lws[0] = max_lws_i;

  gws[1] = (size_t) work_num_item;
  gws[1] *= 5;
  gws[0] = (size_t) grid_points[0]-2;

  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = 1;             threadSize.z = 1;

  CUCHK(hipEventRecord(ev_k_z_solve3_fullopt[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_z_solve3_fullopt",  cmd_q[KERNEL_Q]);
  k_z_solve3_fullopt<<< blockSize, threadSize, 
    sizeof(double)*max_lws_i*(3*5*5+2*5), cmd_q[KERNEL_Q]>>>
      (
       m_rhs[buf_idx], m_lhsA, m_lhsB, m_lhsC,
       grid_points[0], grid_points[1], grid_points[2], work_base, work_num_item, 
       split_flag, work_num_item_default_j
      );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_z_solve3_fullopt",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_z_solve3_fullopt[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  if (timeron) timer_stop(t_zsolve);
}
