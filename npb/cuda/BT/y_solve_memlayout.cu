#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is an CUDA® C version of the NPB BT code. This CUDA® C  //
//  version is a part of SNU-NPB 2019 developed by the Center for Manycore //
//  Programming at Seoul National University and derived from the serial   //
//  Fortran versions in "NPB3.3.1-SER" developed by NAS.                   //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on original NPB 3.3.1, including the technical report, the //
//  original specifications, source code, results and information on how   //
//  to submit new results, is available at:                                //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Information on SNU-NPB 2019, including the conference paper and source //
//  code, is available at:                                                 //
//                                                                         //
//           http://aces.snu.ac.kr                                         //
//                                                                         //
//  Send comments or suggestions for this CUDA® C version to               //
//  snunpb@aces.snu.ac.kr                                                  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 08826, Korea                                             //
//                                                                         //
//          E-mail: snunpb@aces.snu.ac.kr                                  //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Youngdong Do, Hyung Mo Kim, Pyeongseok Oh, Daeyoung Park,      //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include <math.h>
#include <stdio.h>
#include "header.h"

//---------------------------------------------------------------------
// Performs line solves in Y direction by first factoring
// the block-tridiagonal matrix into an upper triangular matrix, 
// and then performing back substitution to solve for the unknow
// vectors of each line.  
// 
// Make sure we treat elements zero to cell_size in the direction
// of the sweep.
//---------------------------------------------------------------------

hipEvent_t   (*ev_k_y_solve_memlayout)[2];

void y_solve_init_memlayout(int iter)
{
  int i;

  ev_k_y_solve_memlayout = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);

  for (i = 0; i < iter; i++) {
    hipEventCreate(&ev_k_y_solve_memlayout[i][0]);

    hipEventCreate(&ev_k_y_solve_memlayout[i][1]);
  }

}

void y_solve_release_memlayout(int iter)
{
  int i;

  for (i = 0; i < iter; i++) {
    hipEventDestroy(ev_k_y_solve_memlayout[i][0]);

    hipEventDestroy(ev_k_y_solve_memlayout[i][1]);
  }

  free(ev_k_y_solve_memlayout);
}

hipEvent_t* y_solve_memlayout(int work_step, 
                               int work_base, 
                               int work_num_item, 
                               int buf_idx)
{
  size_t lws[3];
  size_t gws[3]; 

  if (timeron) timer_start(t_ysolve);

  gws[1] = work_num_item;
  gws[0] = grid_points[0]-2;

  lws[1] = 1;
  lws[0] = min(gws[0], max_work_group_size);

  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  dim3 blockSize(gws[0]/lws[0], gws[1]/lws[1]);
  dim3 threadSize(lws[0], lws[1]);
 
  CUCHK(hipEventRecord(ev_k_y_solve_memlayout[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_y_solve_memlayout",  cmd_q[KERNEL_Q]);
  k_y_solve_memlayout<<<blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_qs, m_rho_i, m_square,
     m_u[buf_idx], m_rhs[buf_idx],
     m_lhs, m_fjac, m_njac,
     grid_points[0], grid_points[1], grid_points[2],
     dy1, dy2, dy3, dy4, dy5,
     c1, c2, ty1, ty2, con43,
     c3c4, c1345, dt,
     work_base, work_num_item, split_flag,
     work_num_item_default
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_y_solve_memlayout",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_y_solve_memlayout[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  if (timeron) timer_stop(t_ysolve);

  return &ev_k_y_solve_memlayout[work_step][1];
}
