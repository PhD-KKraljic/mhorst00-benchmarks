#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is an CUDA® C version of the NPB BT code. This CUDA® C  //
//  version is a part of SNU-NPB 2019 developed by the Center for Manycore //
//  Programming at Seoul National University and derived from the serial   //
//  Fortran versions in "NPB3.3.1-SER" developed by NAS.                   //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on original NPB 3.3.1, including the technical report, the //
//  original specifications, source code, results and information on how   //
//  to submit new results, is available at:                                //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Information on SNU-NPB 2019, including the conference paper and source //
//  code, is available at:                                                 //
//                                                                         //
//           http://aces.snu.ac.kr                                         //
//                                                                         //
//  Send comments or suggestions for this CUDA® C version to               //
//  snunpb@aces.snu.ac.kr                                                  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 08826, Korea                                             //
//                                                                         //
//          E-mail: snunpb@aces.snu.ac.kr                                  //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Youngdong Do, Hyung Mo Kim, Pyeongseok Oh, Daeyoung Park,      //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include <math.h>
#include <stdio.h>
#include "header.h"

//---------------------------------------------------------------------
// 
// Performs line solves in X direction by first factoring
// the block-tridiagonal matrix into an upper triangular matrix, 
// and then performing back substitution to solve for the unknow
// vectors of each line.  
// 
// Make sure we treat elements zero to cell_size in the direction
// of the sweep.
// 
//---------------------------------------------------------------------

hipEvent_t   (*ev_k_x_solve_baseline)[2];

void x_solve_init_baseline(int iter)
{
  int i;

  ev_k_x_solve_baseline = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);

  for (i = 0; i < iter; i++) {
    hipEventCreate(&ev_k_x_solve_baseline[i][0]);

    hipEventCreate(&ev_k_x_solve_baseline[i][1]);
  }
}

void x_solve_release_baseline(int iter)
{
  int i;

  for (i = 0; i < iter; i++) {
    hipEventDestroy(ev_k_x_solve_baseline[i][0]);

    hipEventDestroy(ev_k_x_solve_baseline[i][1]);
  }

  free(ev_k_x_solve_baseline);
}

void x_solve_baseline(int work_step,
                      int work_base,
                      int work_num_item,
                      int buf_idx)
{
  //---------------------------------------------------------------------
  // This function computes the left hand side in the xi-direction
  //---------------------------------------------------------------------

  //---------------------------------------------------------------------
  // determine a (labeled f) and n jacobians
  //---------------------------------------------------------------------

  size_t lws[3];
  size_t gws[3]; 

  if (timeron) timer_start(t_xsolve);

  gws[1] = work_num_item;
  gws[0] = grid_points[1]-2;

  lws[1] = 1;
  lws[0] = min(gws[0], max_work_group_size);

  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  dim3 blockSize(gws[0]/lws[0], gws[1]/lws[1]);
  dim3 threadSize(lws[0], lws[1]);

  CUCHK(hipEventRecord(ev_k_x_solve_baseline[work_step][0], cmd_q[KERNEL_Q]));

  cuda_ProfilerStartEventRecord("k_x_solve_baseline",  cmd_q[KERNEL_Q]);
  k_x_solve_baseline<<<blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_qs, m_rho_i, m_square,
     m_u[buf_idx], m_rhs[buf_idx],
     m_lhs, m_fjac, m_njac,
     grid_points[0], grid_points[1], grid_points[2],
     dx1, dx2, dx3, dx4, dx5,
     c1, c2, tx1, tx2,
     con43, c3c4, c1345, dt,
     work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_x_solve_baseline",  cmd_q[KERNEL_Q]);

  CUCHK(hipEventRecord(ev_k_x_solve_baseline[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  if (timeron) timer_stop(t_xsolve);

}

