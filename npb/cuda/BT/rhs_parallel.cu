//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is an CUDA® C version of the NPB BT code. This CUDA® C  //
//  version is a part of SNU-NPB 2019 developed by the Center for Manycore //
//  Programming at Seoul National University and derived from the serial   //
//  Fortran versions in "NPB3.3.1-SER" developed by NAS.                   //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on original NPB 3.3.1, including the technical report, the //
//  original specifications, source code, results and information on how   //
//  to submit new results, is available at:                                //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Information on SNU-NPB 2019, including the conference paper and source //
//  code, is available at:                                                 //
//                                                                         //
//           http://aces.snu.ac.kr                                         //
//                                                                         //
//  Send comments or suggestions for this CUDA® C version to               //
//  snunpb@aces.snu.ac.kr                                                  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 08826, Korea                                             //
//                                                                         //
//          E-mail: snunpb@aces.snu.ac.kr                                  //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Youngdong Do, Hyung Mo Kim, Pyeongseok Oh, Daeyoung Park,      //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include <math.h>
#include <stdio.h>
#include "header.h"
#include "cuda_util.h"

hipEvent_t   (*ev_k_rhs_datagen_parallel)[2],
              (*ev_k_rhs1_parallel)[2],
              (*ev_k_rhs2_parallel)[2],
              (*ev_k_rhsx1_parallel)[2],
              (*ev_k_rhsx2_parallel)[2],
              (*ev_k_rhsy1_parallel)[2],
              (*ev_k_rhsy2_parallel)[2],
              (*ev_k_rhsz1_parallel)[2],
              (*ev_k_rhsz2_parallel)[2];

void compute_rhs_init_parallel(int iter)
{
  int i;

  ev_k_rhs_datagen_parallel = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhs1_parallel = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhs2_parallel = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhsx1_parallel = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhsx2_parallel = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhsy1_parallel = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhsy2_parallel = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhsz1_parallel = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);
  ev_k_rhsz2_parallel = (hipEvent_t(*)[2])malloc(sizeof(hipEvent_t)*2*iter);

  for (i = 0; i < iter; i++) {
    hipEventCreate(&ev_k_rhs_datagen_parallel[i][0]);
    hipEventCreate(&ev_k_rhs1_parallel[i][0]);
    hipEventCreate(&ev_k_rhs2_parallel[i][0]);
    hipEventCreate(&ev_k_rhsx1_parallel[i][0]);
    hipEventCreate(&ev_k_rhsx2_parallel[i][0]);
    hipEventCreate(&ev_k_rhsy1_parallel[i][0]);
    hipEventCreate(&ev_k_rhsy2_parallel[i][0]);
    hipEventCreate(&ev_k_rhsz1_parallel[i][0]);
    hipEventCreate(&ev_k_rhsz2_parallel[i][0]);

    hipEventCreate(&ev_k_rhs_datagen_parallel[i][1]);
    hipEventCreate(&ev_k_rhs1_parallel[i][1]);
    hipEventCreate(&ev_k_rhs2_parallel[i][1]);
    hipEventCreate(&ev_k_rhsx1_parallel[i][1]);
    hipEventCreate(&ev_k_rhsx2_parallel[i][1]);
    hipEventCreate(&ev_k_rhsy1_parallel[i][1]);
    hipEventCreate(&ev_k_rhsy2_parallel[i][1]);
    hipEventCreate(&ev_k_rhsz1_parallel[i][1]);
    hipEventCreate(&ev_k_rhsz2_parallel[i][1]);
  }
}

void compute_rhs_release_parallel(int iter)
{
  int i;

  for (i = 0; i < iter; i++) {
    hipEventDestroy(ev_k_rhs_datagen_parallel[i][0]);
    hipEventDestroy(ev_k_rhs1_parallel[i][0]);
    hipEventDestroy(ev_k_rhs2_parallel[i][0]);
    hipEventDestroy(ev_k_rhsx1_parallel[i][0]);
    hipEventDestroy(ev_k_rhsx2_parallel[i][0]);
    hipEventDestroy(ev_k_rhsy1_parallel[i][0]);
    hipEventDestroy(ev_k_rhsy2_parallel[i][0]);
    hipEventDestroy(ev_k_rhsz1_parallel[i][0]);
    hipEventDestroy(ev_k_rhsz2_parallel[i][0]);

    hipEventDestroy(ev_k_rhs_datagen_parallel[i][1]);
    hipEventDestroy(ev_k_rhs1_parallel[i][1]);
    hipEventDestroy(ev_k_rhs2_parallel[i][1]);
    hipEventDestroy(ev_k_rhsx1_parallel[i][1]);
    hipEventDestroy(ev_k_rhsx2_parallel[i][1]);
    hipEventDestroy(ev_k_rhsy1_parallel[i][1]);
    hipEventDestroy(ev_k_rhsy2_parallel[i][1]);
    hipEventDestroy(ev_k_rhsz1_parallel[i][1]);
    hipEventDestroy(ev_k_rhsz2_parallel[i][1]);
  }

  free(ev_k_rhs_datagen_parallel);
  free(ev_k_rhs1_parallel);
  free(ev_k_rhs2_parallel);
  free(ev_k_rhsx1_parallel);
  free(ev_k_rhsx2_parallel);
  free(ev_k_rhsy1_parallel);
  free(ev_k_rhsy2_parallel);
  free(ev_k_rhsz1_parallel);
  free(ev_k_rhsz2_parallel);
}

void compute_rhs_body_parallel(int work_step,
                               int work_base,
                               int work_num_item,
                               int copy_buffer_base,
                               int copy_num_item,
                               int buf_idx,
                               hipEvent_t *ev_wb_end_ptr)
{
  size_t lws[3];
  size_t gws[3]; 
  dim3 blockSize;
  dim3 threadSize;


  if (timeron) timer_start(t_rhs);
  lws[2] = 1;
  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[2] = (size_t)copy_num_item;
  gws[1] = (size_t)grid_points[1];
  gws[0] = (size_t)grid_points[0];

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = gws[2]/lws[2]; threadSize.z = lws[2];

  if (buffering_flag && ev_wb_end_ptr != NULL) {
    CUCHK(hipStreamWaitEvent(cmd_q[KERNEL_Q], *ev_wb_end_ptr, 0));
  }

  CUCHK(hipEventRecord(ev_k_rhs_datagen_parallel[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_compute_rhs_data_gen_parallel",  cmd_q[KERNEL_Q]);
  k_compute_rhs_data_gen_parallel<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_rho_i, m_us, m_vs, m_ws, m_qs, m_square, m_u[buf_idx],
     grid_points[0], grid_points[1], grid_points[2],
     copy_buffer_base, copy_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_compute_rhs_data_gen_parallel",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_rhs_datagen_parallel[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  //---------------------------------------------------------------------
  // copy the exact forcing term to the right hand side;  because 
  // this forcing term is known, we can store it on the whole grid
  // including the boundary                   
  //---------------------------------------------------------------------

  lws[2] = 1;
  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[2] = work_num_item;
  gws[1] = (size_t)grid_points[1];
  gws[0] = (size_t)grid_points[0];
  gws[0] *= 5;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = gws[2]/lws[2]; threadSize.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhs1_parallel[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_compute_rhs1_parallel",  cmd_q[KERNEL_Q]);
  k_compute_rhs1_parallel<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_rhs[buf_idx],
     m_forcing[buf_idx],
     grid_points[0], grid_points[1], grid_points[2],
     work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_compute_rhs1_parallel",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_rhs1_parallel[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));





  if (timeron) timer_start(t_rhsx);
  //---------------------------------------------------------------------
  // compute xi-direction fluxes 
  //---------------------------------------------------------------------
  lws[2] = 1;
  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[2] = work_num_item;
  gws[1] = grid_points[1] - 2;
  gws[0] = grid_points[0] - 2;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = gws[2]/lws[2]; threadSize.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhsx1_parallel[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_compute_rhsx1_parallel",  cmd_q[KERNEL_Q]);
  k_compute_rhsx1_parallel<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_us, m_vs, m_ws, m_qs, m_rho_i, m_square,
     m_u[buf_idx],
     m_rhs[buf_idx],
     grid_points[0], grid_points[1], grid_points[2],
     dx1tx1, dx2tx1, dx3tx1, dx4tx1, dx5tx1, xxcon2, xxcon3, xxcon4, xxcon5,
     c1, c2, tx2, con43, dssp, work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_compute_rhsx1_parallel",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_rhsx1_parallel[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  lws[2] = 1;
  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[2] = work_num_item; 
  gws[1] = grid_points[1] - 2;
  gws[0] = grid_points[0] - 2;
  gws[0]*= 5;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = gws[2]/lws[2]; threadSize.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhsx2_parallel[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_compute_rhsx2_parallel",  cmd_q[KERNEL_Q]);
  k_compute_rhsx2_parallel<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_u[buf_idx],
     m_rhs[buf_idx],
     grid_points[0], grid_points[1], grid_points[2],
     dssp, work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_compute_rhsx2_parallel",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_rhsx2_parallel[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));



  if (timeron) timer_stop(t_rhsx);




  if (timeron) timer_start(t_rhsy);

  //---------------------------------------------------------------------
  // compute eta-direction fluxes 
  //---------------------------------------------------------------------
  lws[2] = 1; 
  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[2] = work_num_item; 
  gws[1] = grid_points[1] - 2;
  gws[0] = grid_points[0] - 2;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = gws[2]/lws[2]; threadSize.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhsy1_parallel[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_compute_rhsy1_parallel",  cmd_q[KERNEL_Q]);
  k_compute_rhsy1_parallel<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_us, m_vs, m_ws, m_qs, m_rho_i, m_square,
     m_u[buf_idx], m_rhs[buf_idx],
     grid_points[0], grid_points[1], grid_points[2],
     dy1ty1, dy2ty1, dy3ty1, dy4ty1, dy5ty1, yycon2, yycon3, yycon4, yycon5,
     c1, c2, ty2, con43, dssp, work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_compute_rhsy1_parallel",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_rhsy1_parallel[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  lws[2] = 1;
  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[2] = work_num_item;
  gws[1] = grid_points[1] - 2;
  gws[0] = grid_points[0] - 2;
  gws[0]*= 5;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = gws[2]/lws[2]; threadSize.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhsy2_parallel[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_compute_rhsy2_parallel",  cmd_q[KERNEL_Q]);
  k_compute_rhsy2_parallel<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_u[buf_idx],
     m_rhs[buf_idx],
     grid_points[0], grid_points[1], grid_points[2],
     dssp, work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_compute_rhsy2_parallel",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_rhsy2_parallel[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));





  if (timeron) timer_stop(t_rhsy);


  if (timeron) timer_start(t_rhsz);

  lws[2] = 1;
  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[2] = (size_t)work_num_item;
  gws[1] = (size_t)grid_points[1]-2;
  gws[0] = (size_t)grid_points[0]-2;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = gws[2]/lws[2]; threadSize.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhsz1_parallel[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_compute_rhsz1_parallel",  cmd_q[KERNEL_Q]);
  k_compute_rhsz1_parallel<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_us, m_vs, m_ws, m_qs, m_rho_i, m_square,
     m_u[buf_idx], m_rhs[buf_idx],
     grid_points[0], grid_points[1], grid_points[2],
     dz1tz1, dz2tz1, dz3tz1, dz4tz1, dz5tz1,
     zzcon2, zzcon3, zzcon4, zzcon5, c1, c2,
     tz2, con43, work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_compute_rhsz1_parallel",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_rhsz1_parallel[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));


  lws[2] = 1;
  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[2] = work_num_item;
  gws[1] = grid_points[1]-2;
  gws[0] = grid_points[0]-2;
  gws[0]*= 5;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = gws[2]/lws[2]; threadSize.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhsz2_parallel[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_compute_rhsz2_parallel",  cmd_q[KERNEL_Q]);
  k_compute_rhsz2_parallel<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_u[buf_idx], m_rhs[buf_idx],
     grid_points[0], grid_points[1], grid_points[2], 
     dssp, work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_compute_rhsz2_parallel",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_rhsz2_parallel[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));


  if (timeron) timer_stop(t_rhsz);

  lws[2] = 1;
  lws[1] = 1;
  lws[0] = max_work_group_size;

  gws[2] = work_num_item;
  gws[1] = grid_points[1] - 2;
  gws[0] = grid_points[0] - 2;
  gws[0]*= 5;

  gws[2] = RoundWorkSize(gws[2], lws[2]);
  gws[1] = RoundWorkSize(gws[1], lws[1]);
  gws[0] = RoundWorkSize(gws[0], lws[0]);

  blockSize.x = gws[0]/lws[0]; threadSize.x = lws[0];
  blockSize.y = gws[1]/lws[1]; threadSize.y = lws[1];
  blockSize.z = gws[2]/lws[2]; threadSize.z = lws[2];

  CUCHK(hipEventRecord(ev_k_rhs2_parallel[work_step][0], cmd_q[KERNEL_Q]));
  cuda_ProfilerStartEventRecord("k_compute_rhs2_parallel",  cmd_q[KERNEL_Q]);
  k_compute_rhs2_parallel<<< blockSize, threadSize, 0, cmd_q[KERNEL_Q]>>>
    (
     m_rhs[buf_idx],
     grid_points[0], grid_points[1], grid_points[2],
     dt, work_base, work_num_item, split_flag
    );
  CUCHK(hipGetLastError());
  cuda_ProfilerEndEventRecord("k_compute_rhs2_parallel",  cmd_q[KERNEL_Q]);
  CUCHK(hipEventRecord(ev_k_rhs2_parallel[work_step][1], cmd_q[KERNEL_Q]));

  if (!buffering_flag)
    CUCHK(hipStreamSynchronize(cmd_q[KERNEL_Q]));

  if (timeron) timer_stop(t_rhs);

}

